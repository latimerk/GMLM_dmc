#include "hip/hip_runtime.h"
/*
 * kcGMLMPop_dataStructures.cu
 * Holds all the data - parameters, results, regressors, computation space
 * for a GMLMPop (on one GPU).
 *
 * Package GMLM_dmc for dimensionality reduction of neural data.
 *   
 *  References
 *   Kenneth Latimer & David Freeedman (2021). Low-dimensional encoding of 
 *   decisions in parietal cortex reflects long-term training history.
 *   bioRxiv
 *
 *  Copyright (c) 2021 Kenneth Latimer
 *
 *   This software is distributed under the GNU General Public
 *   License (version 3 or later); please refer to the file
 *   License.txt, included with the software, for details.
 */
#include "kcGMLMPop_dataStructures.hpp"
//#include <tgmath.h>

namespace kCUDA {
   
//for templating the sparse matrix ops
template <class FPTYPE> hipDataType getCudaType();
template <> hipDataType getCudaType<float>() {
return HIP_R_32F;
}
template <> hipDataType getCudaType<double>() {
    return HIP_R_64F;
}  

//============================================================================================================================
//Parameter class
        //constructor
template <class FPTYPE>
GPUGMLMPop_parameters_GPU<FPTYPE>::GPUGMLMPop_parameters_GPU(const GPUGMLMPop_structure_args <FPTYPE> * GMLMPopstructure, const size_t dim_M_, const int dev_, std::shared_ptr<GPUGL_msg> msg_) {
    dev = dev_;
    msg = msg_;
    switchToDevice();
    hipError_t ce;
    hipStream_t stream = 0;

    //setup any log like settings
    logLikeSettings = GMLMPopstructure->logLikeSettings;
    if(GMLMPopstructure->logLikeParams.size() > 0) {
        logLikeParams = new GPUData<FPTYPE>(ce, GPUData_HOST_STANDARD, stream, GMLMPopstructure->logLikeParams.size());
        checkCudaErrors(ce,  "GPUGMLMPop_parameters_GPU errors: could not allocate space for logLikeParams!" );
        for(int ii = 0; ii < GMLMPopstructure->logLikeParams.size(); ii++) {
            (*logLikeParams)[ii] = GMLMPopstructure->logLikeParams[ii];
        }
        ce = logLikeParams->copyHostToGPU(stream);
        checkCudaErrors(ce,  "GPUGMLMPop_parameters_GPU errors: could not copy logLikeParams to GPU!" );
    }
    else {
        logLikeParams = new GPUData<FPTYPE>(ce, GPUData_HOST_NONE, stream, 0);
        checkCudaErrors(ce,  "GPUGMLMPop_parameters_GPU errors: could not allocate space for logLikeParams!" );
    }

    //allocate GPU space for trial weights
    trial_weights_temp = new GPUData<FPTYPE>(ce, GPUData_HOST_PAGELOCKED, stream, dim_M_, GMLMPopstructure->dim_P);
    checkCudaErrors(ce, "GPUGMLMPop_parameters_GPU errors: could not allocate space for trial_weights_temp!" );
    trial_included_temp = new GPUData<unsigned int>(ce, GPUData_HOST_PAGELOCKED, stream, dim_M_, 1);
    checkCudaErrors(ce, "GPUGMLMPop_parameters_GPU errors: could not allocate space for trial_included_temp!" );

    trial_weights_0 = new GPUData<FPTYPE>(ce, GPUData_HOST_NONE, stream, dim_M_, 0);
    checkCudaErrors(ce, "GPUGMLMPop_parameters_GPU errors: could not allocate space for trial_weights_0!" );
    trial_weights = trial_weights_0;

    trial_included_0 = new GPUData<unsigned int>(ce, GPUData_HOST_NONE, stream, dim_M_, 0);
    checkCudaErrors(ce, "GPUGMLMPop_parameters_GPU errors: could not allocate space for trial_included_0!" );
    trial_included = trial_included_0;
    
    //allocate GPU space
    W = new GPUData<FPTYPE>(ce, GPUData_HOST_PAGELOCKED, stream, GMLMPopstructure->dim_P);
    checkCudaErrors(ce, "GPUGMLMPop_parameters_GPU errors: could not allocate space for W!" );
    B = new GPUData<FPTYPE>(ce, GPUData_HOST_PAGELOCKED, stream, GMLMPopstructure->dim_B, GMLMPopstructure->dim_P);
    checkCudaErrors(ce, "GPUGMLMPop_parameters_GPU errors: could not allocate space for B!" );

    //setup each group
    Groups.resize(GMLMPopstructure->Groups.size());
    for(int jj = 0; jj < GMLMPopstructure->Groups.size(); jj++) {
        Groups[jj] = new GPUGMLMPop_parameters_Group_GPU<FPTYPE>(GMLMPopstructure->Groups[jj], this);
    }
}

template <class FPTYPE>
GPUGMLMPop_parameters_Group_GPU<FPTYPE>::GPUGMLMPop_parameters_Group_GPU(const GPUGMLMPop_structure_Group_args<FPTYPE> * GMLMPopGroupStructure, const GPUGMLMPop_parameters_GPU<FPTYPE> * parent_) : parent(parent_) {
    msg = parent->msg;
    dev = parent->dev;
    switchToDevice();
    hipError_t ce;

    hipStream_t stream = 0;

    //allocate GPU space
    V = new GPUData<FPTYPE>(ce, GPUData_HOST_PAGELOCKED, stream, dim_P(), GMLMPopGroupStructure->dim_R_max);
    checkCudaErrors(ce, "GPUGMLMPop_parameters_Group_GPU errors: could not allocate space for V!" );
    T.assign(GMLMPopGroupStructure->dim_S(), NULL);
    F.assign(GMLMPopGroupStructure->dim_D(msg), NULL);
    dF_dT.assign(GMLMPopGroupStructure->dim_S(), NULL);

    compute_dT = new GPUData<bool>(ce, GPUData_HOST_PAGELOCKED, stream, dim_S());
    checkCudaErrors(ce, "GPUGMLMPop_parameters_Group_GPU errors: could not allocate space for compute_dT!" );
    compute_dF = new GPUData<bool>(ce, GPUData_HOST_PAGELOCKED, stream, GMLMPopGroupStructure->dim_D(msg));
    checkCudaErrors(ce, "GPUGMLMPop_parameters_Group_GPU errors: could not allocate space for compute_dF!" );
    
    factor_idx = new GPUData<unsigned int>(ce, GPUData_HOST_STANDARD, stream, dim_S()); 
    checkCudaErrors(ce, "GPUGMLMPop_parameters_Group_GPU errors: could not allocate factor_idx!");
    N_per_factor = new GPUData<unsigned int>(ce, GPUData_HOST_STANDARD, stream, GMLMPopGroupStructure->dim_D(msg)); 
    checkCudaErrors(ce, "GPUGMLMPop_parameters_Group_GPU errors: could not allocate N_per_factor!");

    N_per_factor->assign(0);
    
    std::vector<size_t> dim_F_c;
    dim_F_c.assign(dim_D(), 1);

    dim_F_max   = 0;
    for(int ss = 0; ss < dim_S(); ss++) {
        if(GMLMPopGroupStructure->factor_idx[ss] >= dim_D()) {
            output_stream << "GPUGMLMPop_parameters_Group_GPU errors: invalid factor index!";
            msg->callErrMsgTxt(output_stream);
        }
        (*factor_idx)[ss] = GMLMPopGroupStructure->factor_idx[ss];

        dim_F_c[GMLMPopGroupStructure->factor_idx[ss]] *= GMLMPopGroupStructure->dim_T[ss];
        (*N_per_factor)[GMLMPopGroupStructure->factor_idx[ss]] += 1;
        T[ss] = new GPUData<FPTYPE>(ce, GPUData_HOST_PAGELOCKED, stream, GMLMPopGroupStructure->dim_T[ss], GMLMPopGroupStructure->dim_R_max);
        checkCudaErrors(ce, "GPUGMLMPop_parameters_Group_GPU errors: could not allocate space for T[ss]!" );
    }
    for(int ss = 0; ss < dim_S(); ss++) {
        int dd = (*factor_idx)[ss];
        dF_dT[ss] = new GPUData<FPTYPE>(ce, GPUData_HOST_NONE, stream, GMLMPopGroupStructure->dim_T[ss], dim_F_c[dd], GMLMPopGroupStructure->dim_R_max);
        checkCudaErrors(ce, "GPUGMLMPop_parameters_Group_GPU errors: could not allocate space for dF_dT[ss]!" );
    }
    for(int dd = 0; dd < dim_D(); dd++) {
        if((*N_per_factor)[dd] > 1) {
            F[dd] = new GPUData<FPTYPE>(ce, GPUData_HOST_NONE, stream, dim_F_c[dd], GMLMPopGroupStructure->dim_R_max);
            checkCudaErrors(ce, "GPUGMLMPop_parameters_Group_GPU errors: could not allocate space for F[dd]!" );
        }
        else if((*N_per_factor)[dd] == 1) {
            //find the T for this factor if is unique
            for(int ss = 0; ss < dim_S(); ss++) {
                if((*factor_idx)[ss] == dd) {
                    F[dd] = T[ss];
                    break;
                }
            }
        }
        else {
            output_stream << "GPUGMLMPop_parameters_Group_GPU errors: tensor factor has no components!";
            msg->callErrMsgTxt(output_stream);
        }
        dim_F_max = max(dim_F_max, dim_F_c[dd]);           
    }

    if(GMLMPopGroupStructure->dim_S() == 0 || dim_F_max == 0) {
        output_stream << "GPUGMLMPop_parameters_Group_GPU errors: tensor has no components!";
        msg->callErrMsgTxt(output_stream);
    }

    checkCudaErrors(factor_idx->copyHostToGPU(stream), "GPUGMLMPop_parameters_Group_GPU errors: could not copy factor_idx to device!");
    checkCudaErrors(N_per_factor->copyHostToGPU(stream), "GPUGMLMPop_parameters_Group_GPU errors: could not copy factor_idx to device!"); 
}

//destructor
template <class FPTYPE>
GPUGMLMPop_parameters_GPU<FPTYPE>::~GPUGMLMPop_parameters_GPU() {
    switchToDevice();
    checkCudaErrors("Error in start of GPUGMLMPop_parameters_GPU destructor!");
    
    cudaSafeFree(trial_weights_temp , "GPUGMLMPop_parameters_GPU errors: could not free trial_weights_temp");
    cudaSafeFree(trial_included_temp, "GPUGMLMPop_parameters_GPU errors: could not free trial_included_temp");
    cudaSafeFree(trial_weights_0    , "GPUGMLMPop_parameters_GPU errors: could not free trial_weights_0");
    cudaSafeFree(trial_included_0   , "GPUGMLMPop_parameters_GPU errors: could not free trial_included_0");

    cudaSafeFree(logLikeParams, "GPUGMLMPop_parameters_GPU errors: could not free logLikeParams." );

    cudaSafeFree(W, "GPUGMLMPop_parameters_GPU errors: could not free W");
    cudaSafeFree(B, "GPUGMLMPop_parameters_GPU errors: could not free B");
    for(int jj = 0; jj < Groups.size(); jj++) {
        delete Groups[jj];
    }
}
template <class FPTYPE>
GPUGMLMPop_parameters_Group_GPU<FPTYPE>::~GPUGMLMPop_parameters_Group_GPU() {
    switchToDevice();
    for(int dd = 0; dd < N_per_factor->size(); dd++) {
        if((*N_per_factor)[dd] > 1) {
            cudaSafeFree(F[dd], "GPUGMLMPop_parameters_Group_GPU errors: could not free F");
        }
    }
    cudaSafeFreeVector(T, "GPUGMLMPop_parameters_Group_GPU errors: could not free T");
    cudaSafeFreeVector(dF_dT, "GPUGMLMPop_parameters_Group_GPU errors: could not free dF_dT");
    cudaSafeFree(      V, "GPUGMLMPop_parameters_Group_GPU errors: could not free V");
    cudaSafeFree(compute_dT, "GPUGMLMPop_parameters_Group_GPU errors: could not free compute_dT");
    cudaSafeFree(compute_dF, "GPUGMLMPop_parameters_Group_GPU errors: could not free compute_dF");

    cudaSafeFree(factor_idx, "GPUGMLMPop_parameters_Group_GPU errors: could not free factor_idx");
    cudaSafeFree(N_per_factor, "GPUGMLMPop_parameters_Group_GPU errors: could not free N_per_factor");
}

/* kernel for setting up sparse run indices
*   One thread per trial being run. Sets up a map between the current indices (0:dim_N_temp-1) to the full indices (0:dim_N-1)
* 
*/
__global__ void kernel_ParamsSparseRunSetup(GPUData_kernel<unsigned int> ridx_sa_all,
                                 const GPUData_kernel<unsigned int> trial_included, 
                                 const GPUData_kernel<unsigned int> ridx_st_sall, 
                                 const GPUData_kernel<unsigned int> ridx_t_all,
                                 const GPUData_kernel<size_t> dim_N) {
    unsigned int tr = blockIdx.x * blockDim.x + threadIdx.x;
    if(tr < trial_included.x) {
        unsigned int mm = trial_included[tr];
        unsigned int start_all = ridx_t_all[mm];
        unsigned int start_sp  = ridx_st_sall[tr];
        for(int nn = 0; nn < dim_N[mm]; nn++) {
            ridx_sa_all[nn + start_sp] = start_all + nn;
        }
    }
}


//copy all parameters to GPU
template <class FPTYPE>
void GPUGMLMPop_parameters_GPU<FPTYPE>::copyToGPU(const GPUGMLMPop_params<FPTYPE> * gmlm_params, GPUGMLMPop_dataset_GPU<FPTYPE> * dataset, const hipStream_t stream, const std::vector<hipStream_t> stream_Groups, const GPUGMLMPop_computeOptions<FPTYPE> * opts) {
    switchToDevice();

    //copies trial weights if given
    if(opts != NULL && !opts->trial_weights->empty() && opts->trial_weights->getSize(0) != dataset->max_trials() && (opts->trial_weights->getSize(1) != 1 || opts->trial_weights->getSize(1) != dim_P())) {
        output_stream << "GPUGMLMPop_parameters_GPU errors: input does not have correct number of trial weights" << std::endl;
        msg->callErrMsgTxt(output_stream);
    }
    bool reset_sizes = false;
    if(opts != NULL && opts->trial_weights->size() != 0) {
        size_t trial_weights_nonzero_cnt_c = 0;
        dataset->dim_N_temp = 0;
        checkCudaErrors(trial_weights_temp->resize(stream, -1, opts->trial_weights->getSize(1)), "GPUGMLMPop_parameters_GPU::copyToGPU errors: could not set sizes for sparse run!");
        
        //gets weights for each trial on this GPU block
        for(unsigned int mm = 0; mm < dim_M(); mm++) {
            bool included = false;
            if(opts->trial_weights->getSize(1) == 1) {
                (*trial_weights_temp)[mm] = (*(opts->trial_weights))[(*(dataset->id_t_trial))[mm]];
                included = (*trial_weights_temp)[mm] != 0;
            }
            else {
                for(int pp = 0; pp < dim_P(); pp++) {
                    (*trial_weights_temp)(mm, pp) = (*(opts->trial_weights))((*(dataset->id_t_trial))[mm], pp);
                    included = included || (*trial_weights_temp)(mm,pp) != 0;
                }
            }
    
            //if trial is included
            if(included) {
                (*trial_included_temp)[trial_weights_nonzero_cnt_c] = mm;
                (*(dataset->ridx_st_sall))[trial_weights_nonzero_cnt_c] = dataset->dim_N_temp;
                dataset->dim_N_temp += (*(dataset->dim_N))[mm];
                trial_weights_nonzero_cnt_c++;
            }
        }

        checkCudaErrors(trial_included_temp->resize(stream, trial_weights_nonzero_cnt_c), "GPUGMLMPop_parameters_GPU::copyToGPU errors: could not set sizes for sparse run!");
        trial_included = trial_included_temp;

        if(trial_weights_nonzero_cnt_c != 0) {
            // copies weights to GPU
            checkCudaErrors(trial_weights_temp->copyHostToGPU(stream), "GPUGMLMPop_parameters_GPU errors: could not copy trial_weights_temp to device!");  
            trial_weights = trial_weights_temp;
        }
        else {
            // if there are no trials, might as well not copy anything more
            return;
        }

        //copy list of trials with nonzero weights to host only if the number is small enough for a sparse run
        if(trial_weights_nonzero_cnt_c <= dataset->max_trials_for_sparse_run) {
            //sets some sizes
            checkCudaErrors(dataset->dLL->resize(   stream, dataset->dim_N_temp), "GPUGMLMPop_parameters_GPU::copyToGPU errors: could not set sizes for sparse run!");
            checkCudaErrors(dataset->LL->resize(    stream, dataset->dim_N_temp), "GPUGMLMPop_parameters_GPU::copyToGPU errors: could not set sizes for sparse run!");
            checkCudaErrors(dataset->lambda->resize(stream, dataset->dim_N_temp), "GPUGMLMPop_parameters_GPU::copyToGPU errors: could not set sizes for sparse run!");

            checkCudaErrors(trial_included_temp->resize(  stream, trial_weights_nonzero_cnt_c), "GPUGMLMPop_parameters_GPU::copyToGPU errors: could not set sizes for sparse run!");
            checkCudaErrors(dataset->ridx_st_sall->resize(stream, trial_weights_nonzero_cnt_c), "GPUGMLMPop_parameters_GPU::copyToGPU errors: could not set sizes for sparse run!");
            checkCudaErrors(dataset->ridx_sa_all->resize( stream, dataset->dim_N_temp), "GPUGMLMPop_parameters_GPU::copyToGPU errors: could not set sizes for sparse run!");

            checkCudaErrors(trial_included_temp->copyHostToGPU(  stream), "GPUGMLMPop_parameters_GPU errors: could not copy trial_included_temp to device!");
            checkCudaErrors(dataset->ridx_st_sall->copyHostToGPU(stream), "GPUGMLMPop_parameters_GPU errors: could not copy ridx_st_sall to device!");
            
            trial_included = trial_included_temp;
            dataset->ridx_a_all_c = dataset->ridx_sa_all;
            dataset->ridx_t_all_c = dataset->ridx_st_sall;

            //setup a special index variable
            dim3 block_size;
            block_size.x = min(static_cast<size_t>(1024), trial_weights_nonzero_cnt_c);
            dim3 grid_size;
            grid_size.x = trial_weights_nonzero_cnt_c / block_size.x + ((trial_weights_nonzero_cnt_c % block_size.x == 0)? 0:1);
            kernel_ParamsSparseRunSetup<<<grid_size, block_size,  0, stream>>>(dataset->ridx_sa_all->device(),
                                                                               trial_included->device(), 
                                                                               dataset->ridx_st_sall->device(), 
                                                                               dataset->ridx_t_all->device(),
                                                                               dataset->dim_N->device());
        }
        else {
            reset_sizes = true;
        }
        checkCudaErrors( hipStreamSynchronize(stream), "GPUGMLMPop_parameters_GPU::copyToGPU errors: could not synchronize stream for sparse run!");
    }    
    else {
        // this says all trial weights are 1 (normal log likelihood computation)
        trial_weights  = trial_weights_0;
        reset_sizes = true;
    }

    if(reset_sizes) {
        trial_included = trial_included_0;
        dataset->ridx_a_all_c = dataset->ridx_a_all;
        dataset->ridx_t_all_c = dataset->ridx_t_all;

         //sets some sizes
        checkCudaErrors(dataset->dLL->resize(stream, dataset->dim_N_total()), "GPUGMLM_parameters_GPU::copyToGPU errors: could not set sizes for full run!");
        checkCudaErrors(dataset->LL->resize(stream, dataset->dim_N_total()), "GPUGMLM_parameters_GPU::copyToGPU errors: could not set sizes for full run!");
        checkCudaErrors(dataset->lambda->resize(stream, dataset->dim_N_total()), "GPUGMLM_parameters_GPU::copyToGPU errors: could not set sizes for full run!");
    }
    
    if(gmlm_params != NULL) { //this null check is so I could use this function only to change the weights if I wanted (I probably won't)
        //checks some dimensions
        if(gmlm_params->dim_B() != dim_B()) {
            output_stream << "GPUGMLMPop_parameters_GPU errors: input does not have correct number of linear coefficients (received " << gmlm_params->dim_B() << ", expected " << dim_B() << ")" << std::endl;
            msg->callErrMsgTxt(output_stream);
        }
        if(gmlm_params->dim_P(msg) != dim_P()) {
            output_stream << "GPUGMLMPop_parameters_GPU errors: input does not have correct number of neurons (received " << gmlm_params->dim_P(msg) << ", expected " << dim_P() << ")" << std::endl;
            msg->callErrMsgTxt(output_stream);
        }
        if(opts->Groups.size() != dim_J()) {
            output_stream << "GPUGMLMPop_parameters_GPU errors: input options does not have correct number of groups!" << std::endl;
            msg->callErrMsgTxt(output_stream);
        }

        //copy parameters to GPU
        checkCudaErrors(W->copyTo(stream, gmlm_params->W, false), "GPUGMLMPop_parameters_GPU errors: could not copy W to device!");
        checkCudaErrors(B->copyTo(stream, gmlm_params->B, false), "GPUGMLMPop_parameters_GPU errors: could not copy B to device!");
        
        //for each group
        for(int jj = 0; jj < dim_J(); jj++) {
            Groups[jj]->copyToGPU(gmlm_params->Groups[jj], stream_Groups[jj], opts->Groups[jj]);
        }
    }
}
        
//copy to GPU
template <class FPTYPE>
void GPUGMLMPop_parameters_Group_GPU<FPTYPE>::copyToGPU(const GPUGMLMPop_group_params<FPTYPE> * gmlm_group_params, const hipStream_t stream, const GPUGMLMPop_group_computeOptions * opts) {
    switchToDevice();
    //set current rank
    size_t dim_R_results = gmlm_group_params->dim_R(msg);
    checkCudaErrors(set_dim_R(dim_R_results, stream), "GPUGMLMPop_parameters_Group_GPU errors: could not set new dim_R");

    //check dimensions
    if(dim_S() != gmlm_group_params->dim_S()) {
        output_stream << "GPUGMLMPop_parameters_Group_GPU errors: Invalid tensor coefficient group order. received dim_S = " << gmlm_group_params->dim_S() << ", expected dim_S = " << dim_S() << std::endl;
        msg->callErrMsgTxt(output_stream);
    }
    for(int ss = 0; ss < dim_S(); ss++) {
        if(gmlm_group_params->dim_T(ss, msg) != dim_T(ss)) {
            output_stream << "GPUGMLMPop_parameters_Group_GPU errors: Invalid tensor coefficient size. Received dim_T = " << gmlm_group_params->dim_T(ss, msg) << ", expected dim_T = " << dim_T(ss) << std::endl;
            msg->callErrMsgTxt(output_stream);
        }
    }

    //load compute_T to GPU
    if(opts->compute_dT.size() != dim_S()) {
        output_stream << "GPUGMLMPop_parameters_Group_GPU errors: Invalid compute_dt" << std::endl;
        msg->callErrMsgTxt(output_stream);
    }
    compute_dF->assign(false);
    for(int ss = 0; ss < dim_S(); ss++) {
        (*compute_dT)[ss] = opts->compute_dT[ss];
        (*compute_dF)[(*factor_idx)[ss]] = (*compute_dF)[(*factor_idx)[ss]] || opts->compute_dT[ss];
    }
    checkCudaErrors(compute_dT->copyHostToGPU(stream), "GPUGMLMPop_parameters_Group_GPU errors: could not copy compute_dT to device!");
    checkCudaErrors(compute_dF->copyHostToGPU(stream), "GPUGMLMPop_parameters_Group_GPU errors: could not copy compute_dF to device!");
        
    //copy  to GPU
    checkCudaErrors(V->copyTo(stream, gmlm_group_params->V, false), "GPUGMLMPop_parameters_Group_GPU errors: could not copy V to device!");

    //copy each T
    for(int ss = 0; ss < dim_S(); ss++) {
        checkCudaErrors(T[ss]->copyTo(stream, gmlm_group_params->T[ss], false), "GPUGMLMPop_parameters_Group_GPU errors: could not copy T to device!");
    }

    assembleF(stream);
}

/* kernel for setting up full regressor matrix
* 
*/
        
template <class FPTYPE>
__global__ void kernel_assembleFactorFilter(GPUData_array_kernel<FPTYPE,MAX_DIM_D> F, GPUData_array_kernel<FPTYPE,MAX_DIM_D> dF_dT,
        const GPUData_array_kernel<FPTYPE,MAX_DIM_D> T,
        const GPUData_kernel<unsigned int> factor_idx,
        const GPUData_kernel<unsigned int> N_per_factor, const GPUData_kernel<bool> compute_dT) {
     
    size_t row    = blockIdx.x * blockDim.x + threadIdx.x;
    size_t factor = blockIdx.y * blockDim.y + threadIdx.y;
    
    if(factor < F.N && N_per_factor[factor] > 1) {
        const size_t dim_S = T.N;
        const size_t dim_F = F[factor].x;
        
        if(row < dim_F) {
            for(int rr = 0; rr < F[factor].y; rr++) {
                for(int ss_c = 0; ss_c < dim_S; ss_c++) {
                    if(factor_idx[ss_c] == factor && compute_dT[ss_c]) {
                        for(int tt = 0; tt < dF_dT[ss_c].x; tt++) {
                            dF_dT[ss_c](tt, row, rr) = 0;
                        }
                    }
                }

                size_t T_ctr = 1;
                for(int ss = 0; ss < dim_S; ss++) {
                    if(factor_idx[ss] == factor) {
                        size_t tt = (row/T_ctr) % T[ss].x;
                        if(T_ctr == 1) {
                            F[factor](row, rr)  = T[ss](tt, rr);
                        }
                        else {
                            F[factor](row, rr) *= T[ss](tt, rr);
                        }
                        T_ctr *= T[ss].x;

                        if(compute_dT[ss]) {
                        	dF_dT[ss](tt, row, rr) = 1;
                        }
                    }
                }
                
                for(int ss_c = 0; ss_c < dim_S; ss_c++) {
                    if(factor_idx[ss_c] == factor && compute_dT[ss_c]) {

                        size_t T_ctr = 1;
                        for(int ss = 0; ss < dim_S; ss++) {
                            if(factor_idx[ss] == factor) {
                                size_t tt = (row/T_ctr) % T[ss].x;
                                if(ss != ss_c) {
                                    for(int tt_0 = 0; tt_0 < dF_dT[ss_c].x; tt_0++) {
                                        dF_dT[ss_c](tt_0, row, rr) *= T[ss](tt, rr);
                                    }
                                }
                                T_ctr *= T[ss].x;
                            }
                        }
                    }
                }
            }
        }
    }
}

//assembles the complete regressor matrix (without neuron weights)
//if dim_S>derivative_dim>=0, replaces T[derivative_dim] with ones. if dd < 0, does all factors
template <class FPTYPE>
void GPUGMLMPop_parameters_Group_GPU<FPTYPE>::assembleF(const hipStream_t stream) {
    if(dim_S() > 1) {
        dim3 block_size;
        block_size.x = min(static_cast<size_t>(256), dim_F_max);
        block_size.y = min(static_cast<size_t>(4)  , dim_D());
        dim3 grid_size;
        grid_size.x = dim_F_max / block_size.x + ((dim_F_max % block_size.x == 0)? 0:1);
        grid_size.y = dim_D()   / block_size.y + ((dim_D()   % block_size.y == 0)? 0:1);

        kernel_assembleFactorFilter<<<grid_size, block_size, 0, stream>>>(GPUData<FPTYPE>::assembleKernels(F), GPUData<FPTYPE>::assembleKernels(dF_dT),  GPUData<FPTYPE>::assembleKernels(T), factor_idx->device(), N_per_factor->device(), compute_dT->device());
    }
}

//============================================================================================================================
//Results class
        //constructor
template <class FPTYPE>
GPUGMLMPop_results_GPU<FPTYPE>::GPUGMLMPop_results_GPU(const GPUGMLMPop_structure_args <FPTYPE> * GMLMPopstructure, const size_t max_trials_, const int dev_, std::shared_ptr<GPUGL_msg> msg_) {
    dev = dev_;
    msg = msg_;
    switchToDevice();
    hipError_t ce;
    hipStream_t stream = 0;

    //allocate GPU space for trial weights
    trialLL = new GPUData<FPTYPE>(ce, GPUData_HOST_PAGELOCKED, stream, max_trials_, GMLMPopstructure->dim_P);
    checkCudaErrors(ce, "GPUGMLMPop_results_GPU errors: could not allocate space for trialLL!" );

    //allocate GPU space
    dW = new GPUData<FPTYPE>(ce, GPUData_HOST_PAGELOCKED, stream, GMLMPopstructure->dim_P);
    checkCudaErrors(ce, "GPUGMLMPop_results_GPU errors: could not allocate space for dW!" );
    
    dB = new GPUData<FPTYPE>(ce, GPUData_HOST_PAGELOCKED, stream, GMLMPopstructure->dim_B, GMLMPopstructure->dim_P);
    checkCudaErrors(ce, "GPUGMLMPop_results_GPU errors: could not allocate space for dB!" );

    //setup each group
    Groups.resize(GMLMPopstructure->Groups.size());
    for(int jj = 0; jj < dim_J(); jj++) {
        Groups[jj] = new GPUGMLMPop_results_Group_GPU<FPTYPE>(GMLMPopstructure->Groups[jj], this);
    }
}

template <class FPTYPE>
GPUGMLMPop_results_Group_GPU<FPTYPE>::GPUGMLMPop_results_Group_GPU(const GPUGMLMPop_structure_Group_args<FPTYPE> * GMLMPopGroupStructure, const GPUGMLMPop_results_GPU<FPTYPE> * parent_) : parent(parent_) {
    msg = parent->msg;
    dev = parent->dev;
    switchToDevice();
    hipError_t ce;
    hipStream_t stream = 0;

    //allocate GPU space
    dV = new GPUData<FPTYPE>(ce, GPUData_HOST_PAGELOCKED, stream, dim_P(), GMLMPopGroupStructure->dim_R_max);
    checkCudaErrors(ce, "GPUGMLMPop_results_Group_GPU errors: could not allocate space for dV!" );

    dT.resize(GMLMPopGroupStructure->dim_T.size());
    for(int ss = 0; ss < dim_S(); ss++) {
        dT[ss] = new GPUData<FPTYPE>(ce, GPUData_HOST_PAGELOCKED, stream, GMLMPopGroupStructure->dim_T[ss], GMLMPopGroupStructure->dim_R_max);
        checkCudaErrors(ce, "GPUGMLMPop_results_Group_GPU errors: could not allocate space for T[ss]!" );
    }

    dF.assign(GMLMPopGroupStructure->dim_D(msg), NULL);
    dF_assigned.assign(dim_D(), false);

    std::vector<size_t> dim_F_c;
    std::vector<size_t> NF;
    dim_F_c.assign(dim_D(), 1);
    NF.assign(dim_D(), 0);

    for(int ss = 0; ss < dim_S(); ss++) {
        NF[GMLMPopGroupStructure->factor_idx[ss]]++;
        dim_F_c[GMLMPopGroupStructure->factor_idx[ss]] *= GMLMPopGroupStructure->dim_T[ss];
    }
    
    for(int dd = 0; dd < dim_D(); dd++) {
        if(NF[dd] > 1) {
            dF[dd] = new GPUData<FPTYPE>(ce, GPUData_HOST_NONE, stream, dim_F_c[dd], GMLMPopGroupStructure->dim_R_max);
            checkCudaErrors(ce, "GPUGMLMPop_results_Group_GPU errors: could not allocate space for dF[dd]!" );
            dF_assigned[dd] = true;
        }
        else {
            dF_assigned[dd] = false;
            for(int ss = 0; ss < dim_S(); ss++) {
                if(GMLMPopGroupStructure->factor_idx[ss] == dd) {
                    dF[dd] = dT[ss];
                    break;
                }
            }
        }
    }
}

//destructor
template <class FPTYPE>
GPUGMLMPop_results_GPU<FPTYPE>::~GPUGMLMPop_results_GPU() {
    switchToDevice();
    cudaSafeFree(    trialLL     , "GPUGMLMPop_results_GPU errors: could not free trialLL");

    cudaSafeFree(dW, "GPUGMLMPop_results_GPU errors: could not free W");
    cudaSafeFree(dB, "GPUGMLMPop_results_GPU errors: could not free B");
    for(int jj = 0; jj < Groups.size(); jj++) {
        delete Groups[jj];
    }
}
template <class FPTYPE>
GPUGMLMPop_results_Group_GPU<FPTYPE>::~GPUGMLMPop_results_Group_GPU() {
    switchToDevice();
    cudaSafeFreeVector(dT, "GPUGMLMPop_results_Group_GPU errors: could not free dT");
    for(int dd = 0; dd < dF.size(); dd++) {
        if(dF_assigned[dd]) {
            cudaSafeFree(dF[dd], "GPUGMLMPop_results_Group_GPU errors: could not free dF");
        }
    }
    cudaSafeFree(      dV, "GPUGMLMPop_results_Group_GPU errors: could not free dV");
}

//copy back to host memory (into the object's own page locked memory)
template <class FPTYPE>
void GPUGMLMPop_results_GPU<FPTYPE>::gatherResults(const GPUGMLMPop_parameters_GPU<FPTYPE> * params, const GPUGMLMPop_computeOptions<FPTYPE> * opts, const hipStream_t stream_main, const std::vector<hipStream_t> stream_Groups) {
    switchToDevice();

    //copy the trial-wise log-likelihood
    if(opts->compute_trialLL) {
        checkCudaErrors(trialLL->copyGPUToHost(stream_main), "GPUGMLMPop_results_GPU::copyResultsToHost errors: could not copy trialLL to host!");  
    }

    //copy dW
    if(opts->compute_dW) {
        checkCudaErrors(dW->copyGPUToHost(stream_main), "GPUGMLMPop_results_GPU::copyResultsToHost errors: could not copy dW to host!"); 
    }

    //copy dB
    if(opts->compute_dB) {
        checkCudaErrors(dB->copyGPUToHost(stream_main),"GPUGMLMPop_results_GPU::gatherResults errors: could not copy dB to host!"); 
    }

    //copy each group
    if(opts->Groups.size() != Groups.size()) {
        output_stream << "GPUGMLMPop_Group_GPU::gatherResults errors: invalid options!";
        msg->callErrMsgTxt(output_stream);
    }
    for(int jj = 0; jj < Groups.size(); jj++) {
        Groups[jj]->gatherResults(params->Groups[jj], opts->Groups[jj], stream_Groups[jj]);
    }
}

template <class FPTYPE>
void GPUGMLMPop_results_Group_GPU<FPTYPE>::gatherResults(const GPUGMLMPop_parameters_Group_GPU<FPTYPE> * params, const GPUGMLMPop_group_computeOptions * opts, const hipStream_t stream) {
    switchToDevice();
    //check dims
    if(opts->compute_dT.size() != dT.size()) {
        output_stream << "GPUGMLMPop_results_Group_GPU::gatherResults errors: invalid options!";
        msg->callErrMsgTxt(output_stream);
    }

    //copy dV
    if(opts->compute_dV) {
        checkCudaErrors(dV->copyGPUToHost(stream),"GPUGMLMPop_results_Group_GPU::gatherResults errors: could not copy dV to host!"); 
    }

    //copy dT
    for(int ss = 0; ss < dT.size(); ss++) {
        if(opts->compute_dT[ss]) {
            checkCudaErrors(dT[ss]->copyGPUToHost(stream),"GPUGMLMPop_results_Group_GPU::gatherResults errors: could not copy dT to host!"); 
        }
    }
}

//adds results in page-locked host memory to user-supplied object for returning
template <class FPTYPE>
void GPUGMLMPop_results_GPU<FPTYPE>::addToHost(const GPUGMLMPop_parameters_GPU<FPTYPE> * params, GPUGMLMPop_results<FPTYPE>* results_dest, const GPUGMLMPop_computeOptions<FPTYPE> * opts, const GPUGMLMPop_dataset_GPU<FPTYPE> * dataset, const bool reset) {

    //check the dims of the destination to see if they hold up
    if(opts->compute_trialLL && (results_dest->dim_M() != max_trials() || results_dest->dim_P(msg) != dim_P())) {
        output_stream << "GPUGMLMPop_results_GPU::addResults errors: results.dim_M = " << results_dest->dim_M() << ", "  << results_dest->dim_P(msg) << " is the incorrect size! (expected dim_M = " << max_trials() << ", " << dim_P() << ")";
        msg->callErrMsgTxt(output_stream);
    }
    if(opts->compute_dB && results_dest->dim_B() != dim_B()) {
        output_stream << "GPUGMLMPop_results_GPU::addResults errors: results.dim_B = " << results_dest->dim_B() << " is the incorrect size! (expected dim_B = " << dim_B() << ")";
        msg->callErrMsgTxt(output_stream);
    }
    if(dim_J() != results_dest->Groups.size()) {
        output_stream << "GPUGMLMPop_results_GPU::addResults errors: results.dim_J is the incorrect size!";
        msg->callErrMsgTxt(output_stream);
    }
    
    //if reset, set destination memory to all 0's
    if(reset) {
        if(opts->compute_trialLL) {
            results_dest->trialLL->assign(0);
        }
        if(opts->compute_dW) {
            if(!(dW->isEqualSize(results_dest->dW))) {
                output_stream << "GPUGMLMPop_results_GPU::addResults errors: results.dim_P = " << results_dest->dim_P(msg) << " is the incorrect size! (expected dim_P = " << dim_P() << ")";
                msg->callErrMsgTxt(output_stream);
            }
            results_dest->dW->assign(0);
        }
        if(opts->compute_dB && dim_B() > 0) {
            if(!(dB->isEqualSize(results_dest->dB))) {
                output_stream << "GPUGMLMPop_results_GPU::addResults errors: results.dim_P = " << results_dest->dim_P(msg) << " is the incorrect size! (expected dim_P = " << dim_P() << ")";
                msg->callErrMsgTxt(output_stream);
            }
            results_dest->dB->assign(0);
        }
    }

    //adds local results to dest
    if(opts->compute_trialLL) {
        for(int mm = 0; mm < max_trials(); mm++) {
            if(dataset->isInDataset_trial[mm]) {
                for(int pp = 0; pp < dim_P(); pp++) {
                    FPTYPE weight = 1;
                    if(!opts->trial_weights->empty()) {
                        if(opts->trial_weights->getSize(1) == 1) {
                            weight = (*(opts->trial_weights))[mm];
                        }
                        else {
                            weight = (*(opts->trial_weights))(mm, pp);
                        }
                    }
                    if(weight != 0) {
                        (*(results_dest->trialLL))(mm, pp) += (*trialLL)(mm, pp);
                    }
                }
            }
        }
    }

    if(opts->compute_dW) {
        for(int pp = 0; pp < dim_P(); pp++) {
            (*(results_dest->dW))[pp] += (*dW)[pp];
        }
    }
    if(opts->compute_dB && dim_B() > 0) {
        for(int pp = 0; pp < dim_P(); pp++) {
            for(int bb = 0; bb < dim_B(); bb++) {
                (*(results_dest->dB))(bb, pp) += (*dB)(bb, pp);
            }
        }
    }

    for(int jj = 0; jj < dim_J(); jj++) {
        Groups[jj]->addToHost(params->Groups[jj], results_dest->Groups[jj], opts->Groups[jj], dataset, reset);
    }
}

template <class FPTYPE>
void GPUGMLMPop_results_Group_GPU<FPTYPE>::addToHost(const GPUGMLMPop_parameters_Group_GPU<FPTYPE> * params, GPUGMLMPop_group_results<FPTYPE>* results_dest, const GPUGMLMPop_group_computeOptions * opts, const GPUGMLMPop_dataset_GPU<FPTYPE> * dataset, const bool reset) {
    //check the dims of the destination to see if they hold up
    //if reset, set destination memory to all 0's
    if(reset) {
        if(opts->compute_dV) {
            if(!(dV->isEqualSize(results_dest->dV))) {
                output_stream << "GPUGMLMPop_results_Group_GPU::addResults errors: results struct is the incorrect size!";
                msg->callErrMsgTxt(output_stream);
            }
            results_dest->dV->assign(0);
        }
        if(results_dest->dim_S() != dim_S()) {
            output_stream << "GPUGMLMPop_results_Group_GPU::addResults errors: results struct is the incorrect size!";
            msg->callErrMsgTxt(output_stream);
        }
        for(int ss = 0; ss < dim_S(); ss++) {
            if(opts->compute_dT[ss]) {
                if(!(dT[ss]->isEqualSize(results_dest->dT[ss]))) {
                    output_stream << "GPUGMLMPop_results_Group_GPU::addResults errors: results struct is the incorrect size!";
                    msg->callErrMsgTxt(output_stream);
                }
                results_dest->dT[ss]->assign(0);
            }
        }
    }

    //adds on results
    if(opts->compute_dV) {
        for(int pp = 0; pp < parent->dim_P(); pp++) {
            for(int rr = 0; rr < dim_R(); rr++) {
                (*(results_dest->dV))(pp, rr) += (*dV)(pp, rr);
            }
        }
    }

    for(int ss = 0; ss < dim_S(); ss++) {
        if(opts->compute_dT[ss]) {
            for(int tt = 0; tt < dim_T(ss); tt++) {
                for(int rr = 0; rr < dim_R(); rr++) {
                    (*(results_dest->dT[ss]))(tt, rr) += (*(dT[ss]))(tt, rr);
                }
            }
        }
    }
}

//============================================================================================================================
//Dataset class
        
//Constructor takes in all the group data and GMLMPop setup
template <class FPTYPE>
GPUGMLMPop_dataset_GPU<FPTYPE>::GPUGMLMPop_dataset_GPU(const GPUGMLMPop_structure_args<FPTYPE> * GMLMPopstructure, const GPUGMLMPop_GPU_block_args <FPTYPE> * block, const size_t max_trials_, const hipStream_t stream, const std::vector<hipsparseHandle_t> & cusparseHandle_Groups, std::shared_ptr<GPUGL_msg> msg_) {
    dev = block->dev_num;
    msg = msg_;
    switchToDevice();
    hipError_t ce;

    log_dt = log(GMLMPopstructure->binSize);
    Groups.assign(GMLMPopstructure->Groups.size(), NULL); //sets up dim_J()
    dim_N = new GPUData<size_t>(ce, GPUData_HOST_STANDARD, stream, block->trials.size()); //sets up dim_M()
    checkCudaErrors(ce, "GPUGMLMPop_dataset_GPU errors: could not allocate dim_N on device!");
            
    // number of trials
    isInDataset_trial.assign( max_trials_, false); //if each trial is in this block
    if(dim_M() == 0) {
        output_stream << "GPUGMLMPop_dataset_GPU errors: no trials given to GPU block!";
        msg->callErrMsgTxt(output_stream);
    }

    max_trials_for_sparse_run = min(dim_M()/2, static_cast<size_t>(block->max_trials_for_sparse_run));

    // setup up the order that trials go to the GPU
    //   in blocks ordered by neurons     

    size_t dim_N_total_c = 0;
    dim_N_temp = 0;
    max_trial_length = 1;

    ridx_t_all = new GPUData<unsigned int>(ce, GPUData_HOST_STANDARD, stream, dim_M());
    checkCudaErrors(ce, "GPUGMLMPop_dataset_GPU errors: could not allocate ridx_t_all on device!");
    id_t_trial = new GPUData<unsigned int>(ce, GPUData_HOST_STANDARD, stream, dim_M());
    checkCudaErrors(ce, "GPUGMLMPop_dataset_GPU errors: could not allocate id_t_trial on device!");
    normalizingConstants_trial = new GPUData<FPTYPE>(ce, GPUData_HOST_STANDARD, stream, dim_M(), GMLMPopstructure->dim_P);
    checkCudaErrors(ce, "GPUGMLMPop_dataset_GPU errors: could not allocate normalizingConstants_trial on device!");

    size_t X_lin_depth;
    for(int mm = 0; mm < dim_M(); mm++) {
        if(mm == 0) {
            X_lin_depth = block->trials[mm]->X_lin->getSize(2);
        }

        if(block->trials[mm]->X_lin->getSize(2) != X_lin_depth || (X_lin_depth > 1 && X_lin_depth != GMLMPopstructure->dim_P)) {
            output_stream << "GPUGMLMPop_dataset_GPU errors: invalid size of X_lin: depth must be 0-1 or dim_P!";
            msg->callErrMsgTxt(output_stream);
        }
        
        //save trial indices
        (*ridx_t_all)[mm] = dim_N_total_c;

        // get trial length
        (*dim_N)[mm] = block->trials[mm]->dim_N();
        if((*dim_N)[mm] == 0) {
            output_stream << "GPUGMLMPop_dataset_GPU errors: trials cannot be empty!";
            msg->callErrMsgTxt(output_stream);
        }
        dim_N_total_c += (*dim_N)[mm]; // add length to total 

        max_trial_length = max(max_trial_length, (*dim_N)[mm]); //update max trial length

        //save trial and neuron number
        (*id_t_trial)[mm] = block->trials[mm]->trial_idx;
        if(isInDataset_trial[block->trials[mm]->trial_idx]) { //trial index already found
            output_stream << "GPUGMLMPop_dataset_GPU errors: trial indices must be unique!";
            msg->callErrMsgTxt(output_stream);
        }

        isInDataset_trial[block->trials[mm]->trial_idx] = true;

        for(int pp = 0; pp < GMLMPopstructure->dim_P; pp++) {
            FPTYPE nc = 0; // normalizing constant
            for(int nn = 0; nn < (*dim_N)[mm]; nn++) {
                if(GMLMPopstructure->logLikeSettings == ll_poissExp) {
                    FPTYPE Y_c = (*(block->trials[mm]->Y))(nn, pp);
                    nc += (Y_c >= 0) ? -lgamma(floor(Y_c) + 1.0) : 0;
                }
            }
            (*normalizingConstants_trial)(mm, pp) = nc;
        }
    }
    id_a_trialM = new GPUData<unsigned int>(ce, GPUData_HOST_STANDARD, stream, dim_N_total_c);
    checkCudaErrors(ce, "GPUGMLMPop_dataset_GPU errors: could not allocate id_a_trialM on device!");

    size_t N_total_ctr = 0;
    for(int mm = 0; mm < dim_M(); mm++) {
        for(int nn = 0; nn < (*dim_N)[mm]; nn++) {
            (*id_a_trialM)[N_total_ctr + nn] = mm;
        }
        N_total_ctr += (*dim_N)[mm];
    }

    //allocate space on GPU for data and copy any local values to GPU
        //spike counts
    Y = new GPUData<FPTYPE>(ce, GPUData_HOST_NONE, stream, dim_N_total_c, GMLMPopstructure->dim_P);
    checkCudaErrors(ce, "GPUGMLMPop_dataset_GPU errors: could not allocate Y on device!");
    
        //linear term (divded up into per-neuron blocks)
    X_lin = new GPUData<FPTYPE>(ce, GPUData_HOST_NONE, stream, dim_N_total_c, GMLMPopstructure->dim_B, X_lin_depth);
    checkCudaErrors(ce, "GPUGMLMPop_dataset_GPU errors: could not allocate X_lin on device!");
        
        //copy each trial to GPU
    for(int mm = 0; mm < dim_M(); mm++) {
        // spike counts
        hipPos copyOffset = make_hipPos((*ridx_t_all)[mm], 0, 0);
        checkCudaErrors(Y->copyTo(stream, block->trials[mm]->Y, true, copyOffset), "GPUGMLMPop_dataset_GPU errors: could not copy Y to device!");
                
        // linear term
        if(!X_lin->empty()) { //don't call if no linear term
            checkCudaErrors(X_lin->copyTo(stream, block->trials[mm]->X_lin, true, copyOffset), "GPUGMLMPop_dataset_GPU errors: could not copy X_lin to device!");
        }
    } 

    //upload vectors to GPU
    checkCudaErrors(normalizingConstants_trial->copyHostToGPU(stream), "GPUGMLMPop_dataset_GPU errors: could not copy normalizingConstants_trial to device!");
   
    checkCudaErrors(ridx_t_all->copyHostToGPU(stream), "GPUGMLMPop_dataset_GPU errors: could not copy ridx_t_all to device!");
    checkCudaErrors(id_t_trial->copyHostToGPU(stream), "GPUGMLMPop_dataset_GPU errors: could not copy id_t_trial to device!");
    checkCudaErrors(id_a_trialM->copyHostToGPU(stream), "GPUGMLMPop_dataset_GPU errors: could not copy id_a_trialM to device!");
     
    checkCudaErrors(dim_N->copyHostToGPU(stream), "GPUGMLMPop_dataset_GPU errors: could not copy dim_N to device!");

    //setup compute space
     LL = new GPUData<FPTYPE>(ce, GPUData_HOST_NONE, stream, dim_N_total(), dim_P());
    checkCudaErrors(ce, "GPUGMLMPop_dataset_GPU errors: could not allocate LL on device!");
    dLL = new GPUData<FPTYPE>(ce, GPUData_HOST_NONE, stream, dim_N_total(), dim_P());
    checkCudaErrors(ce, "GPUGMLMPop_dataset_GPU errors: could not allocate dLL on device!");

    ridx_sa_all = new GPUData<unsigned int>(ce, GPUData_HOST_NONE, stream, dim_N_total());
    checkCudaErrors(ce, "GPUGMLMPop_dataset_GPU errors: could not allocate ridx_sa_all on device!");
    ridx_a_all = new GPUData<unsigned int>(ce, GPUData_HOST_NONE, stream, dim_N_total(), 0);
    checkCudaErrors(ce, "GPUGMLMPop_dataset_GPU errors: could not allocate ridx_a_all on device!");
    ridx_st_sall = new GPUData<unsigned int>(ce, GPUData_HOST_PAGELOCKED, stream, dim_M());
    checkCudaErrors(ce, "GPUGMLMPop_dataset_GPU errors: could not allocate ridx_st_sall on device!");
    
    lambda = new GPUData<FPTYPE>(ce, GPUData_HOST_NONE, stream, dim_N_total(), dim_P(), dim_J());
    checkCudaErrors(ce, "GPUGMLMPop_dataset_GPU errors: could not allocate lambda on device!");

    X_lin_temp = new GPUData<FPTYPE>(ce, GPUData_HOST_NONE, stream, max_trial_length * max_trials_for_sparse_run, dim_B(), X_lin_depth);
    checkCudaErrors(ce, "GPUGMLMPop_dataset_GPU errors: could not allocate X_lin_temp on device!");

    dW_trial = new GPUData<FPTYPE>(ce, GPUData_HOST_NONE, stream, dim_M(), dim_P());
    checkCudaErrors(ce, "GPUGMLMPop_dataset_GPU errors: could not allocate dW_trial on device!");

    //setup the groups
    for(int jj = 0; jj < dim_J(); jj++) {
        Groups[jj] = new GPUGMLMPop_dataset_Group_GPU<FPTYPE>(jj, GMLMPopstructure->Groups[jj], block->trials, this, stream, cusparseHandle_Groups[jj]);
    }
}

template <class FPTYPE>
GPUGMLMPop_dataset_Group_GPU<FPTYPE>::GPUGMLMPop_dataset_Group_GPU(const int groupNum_, const GPUGMLMPop_structure_Group_args<FPTYPE> * GMLMPopGroupStructure, const std::vector<GPUGMLMPop_trial_args <FPTYPE> *> trials, const GPUGMLMPop_dataset_GPU<FPTYPE> * parent_, const hipStream_t stream, const hipsparseHandle_t & cusparseHandle) : parent(parent_), groupNum(groupNum_) {
    dev = parent->dev;
    msg = parent->msg;
    switchToDevice();
    hipError_t ce;
    
    //sets up dimensions
    X.resize( GMLMPopGroupStructure->dim_D(msg));
    XF.resize(dim_D());
    iX.resize(dim_D());
    X_temp.resize( dim_D());
    lambda_d.resize( dim_D());

    isShared = new GPUData<bool>(ce, GPUData_HOST_STANDARD, stream, dim_D()); 
    checkCudaErrors(ce, "GPUGMLMPop_dataset_Group_GPU errors: could not allocate isShared!");
    isSharedIdentity = new GPUData<bool>(ce, GPUData_HOST_STANDARD, stream, dim_D()); 
    checkCudaErrors(ce, "GPUGMLMPop_dataset_Group_GPU errors: could not allocate isSharedIdentity!");

    dim_A = GMLMPopGroupStructure->dim_A;

    size_t dim_T_total = 1;
    std::vector<size_t> dim_F_c;
    dim_F_c.assign(dim_D(), 1);
    for(int ss = 0; ss < GMLMPopGroupStructure->dim_S(); ss++) {
        dim_T_total *= GMLMPopGroupStructure->dim_T[ss];
        dim_F_c[GMLMPopGroupStructure->factor_idx[ss]] *= GMLMPopGroupStructure->dim_T[ss];
    }

    if(GMLMPopGroupStructure->dim_S() == 0 || dim_T_total == 0) {
        output_stream << "GPUGMLMPop_dataset_Group_GPU errors: tensor has no components!";
        msg->callErrMsgTxt(output_stream);
    }
    if(GMLMPopGroupStructure->dim_A == 0) {
        output_stream << "GPUGMLMPop_dataset_Group_GPU errors: tensor has no events/data!";
        msg->callErrMsgTxt(output_stream);
    }
    
    //allocated space for regressors and copy to GPU
    size_t max_dim_X_shared = parent->dim_N_total();

    for(int dd = 0; dd < dim_D(); dd++) {
        (*isShared)[dd] = !(GMLMPopGroupStructure->X_shared[dd]->empty());

        if((*isShared)[dd]) {
            //if shared
            max_dim_X_shared = max(max_dim_X_shared, GMLMPopGroupStructure->X_shared[dd]->getSize(0));

            //gets depth
            size_t depth = GMLMPopGroupStructure->X_shared[dd]->getSize(2);
            if(depth != 1) {
                output_stream << "GPUGMLMPop_dataset_Group_GPU errors: X_shared depth must be 1!";
                msg->callErrMsgTxt(output_stream);
            }

            //allocate space
            X[dd]  = new GPUData<FPTYPE>(ce, GPUData_HOST_NONE, stream, GMLMPopGroupStructure->X_shared[dd]->getSize(0), dim_F_c[dd], depth);
            iX[dd] = new GPUData<int   >(ce, GPUData_HOST_NONE, stream, parent->dim_N_total(), dim_A);

            //copy to GPU
            checkCudaErrors(X[dd]->copyTo(stream, GMLMPopGroupStructure->X_shared[dd], false), "GPUGMLMPop_dataset_Group_GPU errors: could not copy X[dd] shared to device!");
       
            // copy each trial's data to GPU
            for(int mm = 0; mm < trials.size(); mm++) {
                hipPos copyOffset = make_hipPos((*(parent->ridx_t_all))[mm], 0, 0); //get row for current trial
                checkCudaErrors(iX[dd]->copyTo(stream, trials[mm]->Groups[groupNum]->iX[dd], true, copyOffset), "GPUGMLMPop_dataset_Group_GPU errors: could not copy iX[dd] shared to device!");
            }

            //check if X_shared is the identity matrix
            if(X[dd]->getSize(0) == X[dd]->getSize(1)) {
                (*isSharedIdentity)[dd] = true;
                for(int ii = 0; ii < X[dd]->getSize(0) && (*isSharedIdentity)[dd]; ii++) {
                    for(int jj = 0; jj < X[dd]->getSize(1) && (*isSharedIdentity)[dd]; jj++) {
                        if(ii == jj) {
                            (*isSharedIdentity)[dd] = 1 == (*(GMLMPopGroupStructure->X_shared[dd]))(ii,jj);
                        }
                        else {
                            (*isSharedIdentity)[dd] = 0 == (*(GMLMPopGroupStructure->X_shared[dd]))(ii,jj);
                        }
                    }
                }
            }
            else {
                (*isSharedIdentity)[dd] = false;
            }

            if(!((*isSharedIdentity)[dd])) {
                //XF comp space
                XF[dd] = new GPUData<FPTYPE>(ce, GPUData_HOST_NONE, stream, dim_X(dd), GMLMPopGroupStructure->dim_R_max);
                checkCudaErrors(ce, "GPUGMLMPop_dataset_Group_GPU errors: could not allocate space for XF[dd] shared!" );
            }
            else {
                XF[dd] = new GPUData<FPTYPE>(ce, GPUData_HOST_NONE, stream, dim_X(dd), GMLMPopGroupStructure->dim_R_max, 0); // is empty, but has correct dimensions
                checkCudaErrors(ce, "GPUGMLMPop_dataset_Group_GPU errors: could not allocate space for XF[dd] shared+identity!" );
            }

            //X space for sparse runs
            X_temp[dd] = new GPUData<FPTYPE>(ce, GPUData_HOST_NONE, stream, parent->max_trial_length * parent->max_trials_for_sparse_run, dim_F_c[dd], dim_A, true);
            checkCudaErrors(ce, "GPUGMLMPop_dataset_Group_GPU errors: could not allocate space for X_temp[dd]!" );
        }
        else {
            //if local
            (*isSharedIdentity)[dd] = false;

            //gets depth
            size_t depth = trials[0]->Groups[groupNum]->X[dd]->getSize(2);
            if(depth != 1 && depth != dim_A) {
                output_stream << "GPUGMLMPop_dataset_Group_GPU errors: X_local depth must be dim_A or 1!";
                msg->callErrMsgTxt(output_stream);
            }

            //allocate space
            X[dd]  = new GPUData<FPTYPE>(ce, GPUData_HOST_NONE, stream, parent->dim_N_total(), dim_F_c[dd], depth, true);
            iX[dd] = new GPUData<int   >(ce, GPUData_HOST_NONE, stream, 0, GMLMPopGroupStructure->dim_A);

            // copy each trial's data
            for(int mm = 0; mm < trials.size(); mm++) {
                hipPos copyOffset = make_hipPos((*(parent->ridx_t_all))[mm], 0, 0); //get row for current trial
                checkCudaErrors(X[dd]->copyTo(stream, trials[mm]->Groups[groupNum]->X[dd], true, copyOffset), "GPUGMLMPop_dataset_Group_GPU errors: could not copy X[dd] local to device!");
            }

            //XF comp space
            XF[dd] = new GPUData<FPTYPE>(ce, GPUData_HOST_NONE, stream, parent->dim_N_total(), GMLMPopGroupStructure->dim_R_max, depth, true);
            checkCudaErrors(ce, "GPUGMLMPop_dataset_Group_GPU errors: could not allocate space for XF[dd] shared!" );

            //X space for sparse runs
            X_temp[dd] = new GPUData<FPTYPE>(ce, GPUData_HOST_NONE, stream, parent->max_trial_length * parent->max_trials_for_sparse_run, dim_F_c[dd], depth, true);
            checkCudaErrors(ce, "GPUGMLMPop_dataset_Group_GPU errors: could not allocate space for X_temp[dd]!" );
        }

    }

    checkCudaErrors(isShared->copyHostToGPU(stream), "GPUGMLMPop_dataset_Group_GPU errors: could not copy isShared to device!");
    checkCudaErrors(isSharedIdentity->copyHostToGPU(stream), "GPUGMLMPop_dataset_Group_GPU errors: could not copy isSharedIdentity to device!");
    
    //setup compute space
    lambda_v = new GPUData<FPTYPE>(ce, GPUData_HOST_NONE, stream, parent->dim_N_total(), dim_R_max());
    checkCudaErrors(ce, "GPUGMLMPop_dataset_Group_GPU errors: could not allocate space for lambda_v!" );

    // pitched memory for lambda_d: note arrangement is (dim_N_total*dim_A) x dim_R
    //                                this stacks the events to line up with X or S
    lambda_d.assign(dim_D(), NULL);
    for(int dd = 0; dd < dim_D(); dd++) {
        size_t depth = dim_A;
        if(!((*isShared)[dd]) && X[dd]->getSize(2) == 1) {
            depth = 1;
        }
        lambda_d[dd] = new GPUData<FPTYPE>(ce, GPUData_HOST_NONE, stream, parent->dim_N_total(), dim_R_max(), depth, true);
        checkCudaErrors(ce, "GPUGMLMPop_dataset_Group_GPU errors: could not allocate space for lambda_d!" );
    }

    phi_d =  new GPUData<FPTYPE>(ce, GPUData_HOST_NONE, stream, max_dim_X_shared, dim_R_max());
    checkCudaErrors(ce, "GPUGMLMPop_dataset_Group_GPU errors: could not allocate space for phi_d!" );

    //setup sparse matrices for dT
    spi_rows.assign(dim_D(), NULL);
    spi_cols.assign(dim_D(), NULL);
    spi_data.assign(dim_D(), NULL);

    spi_S.assign(dim_D(), NULL);
    spi_phi_d.assign(dim_D(), NULL);
    spi_lambda_d.assign(dim_D(), NULL);

    spi_buffer.assign(dim_D(), NULL);
    spi_buffer_size.assign(dim_D(), 0);

    for(int dd = 0; dd < dim_D(); dd++) {
        if((*isShared)[dd]) {
            //gets the rows and cols of the spm in the correct order
                //shorter algorithm is too slow for my level of patience, so we do this in a couple steps
                //first, get valid entries and number of entries per row of spi_S
            size_t ctr = 0;
            std::vector<int> row_ctr;
            row_ctr.resize(dim_X(dd));
            for(int mm = 0; mm < parent->dim_M(); mm++) { //for each trial
                for(int aa = 0; aa < dim_A; aa++) { //for each event
                    for(int nn = 0; nn < trials[mm]->dim_N(); nn++) { //for each observation
                        //gets the entry in the input data
                        int row = (*(trials[mm]->Groups[groupNum]->iX[dd]))(nn, aa);
                        if(row >= 0 && row < dim_X(dd)) { //if valid row (invalid indices are 0's)
                            row_ctr[row]++;
                            ctr++;
                        }
                    }
                }
            }

                //gets to cumulative sum of the rows
            std::vector<int> row_idx;
            row_idx.resize(dim_X(dd));
            row_idx[0] = 0;
            for(int xx = 1; xx < dim_X(dd); xx++) {
                row_idx[xx] = row_ctr[xx-1] + row_idx[xx-1]; 
            }
                //goes back through the indices and adds them on
            spi_rows[dd] = new GPUData<int>(ce, GPUData_HOST_STANDARD, stream, ctr, 1, 1);
            checkCudaErrors(ce, "GPUGMLMPop_dataset_Group_GPU errors: could not allocate space for spi_rows[dd]!");
            spi_cols[dd] = new GPUData<int>(ce, GPUData_HOST_STANDARD, stream, ctr, 1, 1);
            checkCudaErrors(ce, "GPUGMLMPop_dataset_Group_GPU errors: could not allocate space for spi_cols[dd]!");

            row_ctr.assign(dim_X(dd), 0); //reset row counter
            for(int mm = 0; mm < parent->dim_M(); mm++) { //for each trial
                for(int aa = 0; aa < dim_A; aa++) { //for each event
                    for(int nn = 0; nn < trials[mm]->dim_N(); nn++) { //for each observation
                        //gets the entry in the input data
                        int row = (*(trials[mm]->Groups[groupNum]->iX[dd]))(nn, aa);
                        if(row >= 0 && row < dim_X(dd)) { //if valid row
                            //inserts element
                            size_t entry_num = row_idx[row] + row_ctr[row];
                            (*(spi_cols[dd]))[entry_num] = (*(parent->ridx_t_all))[mm] + nn + aa * parent->dim_N_total();
                            (*(spi_rows[dd]))[entry_num] = row;

                            row_ctr[row]++;
                        }
                    }
                }
            }

            //copy indices to device
            checkCudaErrors(spi_rows[dd]->copyHostToGPU(stream), "GPUGMLMPop_dataset_Group_GPU errors: could not copy spi_rows[dd] to device!");
            checkCudaErrors(spi_cols[dd]->copyHostToGPU(stream), "GPUGMLMPop_dataset_Group_GPU errors: could not copy spi_cols[dd] to device!");
            
            spi_data[dd] = new GPUData<FPTYPE>(ce, GPUData_HOST_NONE, stream, spi_rows[dd]->size(), 1, 1);
            checkCudaErrors(ce, "GPUGMLMPop_dataset_Group_GPU errors: could not allocate space for spi_data[dd]!");

            //setup sparse matrix handle
            hipsparseStatus_t cusparse_stat;
            spi_S[dd] =  new hipsparseSpMatDescr_t;
            cusparse_stat = hipsparseCreateCoo(spi_S[dd],
                        dim_X(dd), lambda_d[dd]->getSize(0) * lambda_d[dd]->getSize(2), //num rows, cols
                        spi_rows[dd]->size(), //number of non-zeros
                        spi_rows[dd]->getData_gpu(), //row offsets
                        spi_cols[dd]->getData_gpu(), //col offsets
                        spi_data[dd]->getData_gpu(), //the entries
                        HIPSPARSE_INDEX_32I,
                        HIPSPARSE_INDEX_BASE_ZERO,
                        getCudaType<FPTYPE>());
            checkCudaErrors(cusparse_stat, "GPUGMLMPop_dataset_Group_GPU errors: creating sparse mat spi_S for dT failed.");

            //setup dense handle for lambda_d
            spi_lambda_d[dd] = new hipsparseDnVecDescr_t;
            cusparse_stat = hipsparseCreateDnVec(spi_lambda_d[dd],
                                                lambda_d[dd]->getSize(0) * lambda_d[dd]->getSize(2),  //size
                                                lambda_d[dd]->getData_gpu(),
                                                getCudaType<FPTYPE>());
            checkCudaErrors(cusparse_stat, "GPUGMLMPop_dataset_Group_GPU errors: creating dense vec cusparse handle spi_lambda_d failed.");

            //setup dense handle for phi_d
            spi_phi_d[dd] = new hipsparseDnVecDescr_t;
            cusparse_stat = hipsparseCreateDnVec(spi_phi_d[dd],
                                                dim_X(dd), //size
                                                phi_d->getData_gpu(), //values
                                                getCudaType<FPTYPE>()); //valueType
            checkCudaErrors(cusparse_stat, "GPUGMLMPop_dataset_Group_GPU errors: creating dense vec cusparse handle spi_phi_d failed.");

            //checks buffer for spi
            size_t buffer;
            FPTYPE alpha = 1;
            FPTYPE beta  = 0;
            cusparse_stat = hipsparseSpMV_bufferSize(cusparseHandle,
                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                    &alpha,
                    *(spi_S[dd]),
                    *(spi_lambda_d[dd]),
                    &beta,
                    *( spi_phi_d[dd] ),
                    getCudaType<FPTYPE>(),
                    HIPSPARSE_SPMV_ALG_DEFAULT,
                    &(buffer));
            checkCudaErrors(cusparse_stat, "GPUGMLMPop_dataset_Group_GPU errors: getting buffer size for SpMV failed.");

            spi_buffer[dd] = new GPUData<char>(ce, GPUData_HOST_NONE, stream, buffer, 1, 1);
            checkCudaErrors(ce, "GPUGMLMPop_dataset_Group_GPU errors: could not allocate space for spi_buffer[dd]!" );
            spi_buffer_size[dd] = buffer; 
        }
    }
}

// destructor
template <class FPTYPE>
GPUGMLMPop_dataset_GPU<FPTYPE>::~GPUGMLMPop_dataset_GPU() {
    cudaSafeFree(Y, "GPUGMLMPop_dataset_GPU errors: could not free Y");
    
    cudaSafeFree(X_lin, "GPUGMLMPop_dataset_GPU errors: could not free X_lin");
    cudaSafeFree(X_lin_temp, "GPUGMLMPop_dataset_GPU errors: could not free X_lin_temp");
    
    cudaSafeFree(normalizingConstants_trial, "GPUGMLMPop_dataset_GPU errors: could not free normalizingConstants_trial");
    
    cudaSafeFree(ridx_t_all   , "GPUGMLMPop_dataset_GPU errors: could not free ridx_t_all");
    cudaSafeFree(ridx_st_sall , "GPUGMLMPop_dataset_GPU errors: could not free ridx_st_sall");
    cudaSafeFree(ridx_sa_all  , "GPUGMLMPop_dataset_GPU errors: could not free ridx_sa_all");
    cudaSafeFree(ridx_a_all  , "GPUGMLMPop_dataset_GPU errors: could not free ridx_a_all");
    
    cudaSafeFree(id_t_trial , "GPUGMLMPop_dataset_GPU errors: could not free id_t_trial");
    cudaSafeFree(id_a_trialM, "GPUGMLMPop_dataset_GPU errors: could not free id_a_trialM");
    
    cudaSafeFree(dim_N, "GPUGMLMPop_dataset_GPU errors: could not free dim_N");
    
    cudaSafeFree( LL, "GPUGMLMPop_dataset_GPU errors: could not free  LL");
    cudaSafeFree(dLL, "GPUGMLMPop_dataset_GPU errors: could not free dLL");
    cudaSafeFree(lambda, "GPUGMLMPop_dataset_GPU errors: could not free lambda");
    cudaSafeFree(dW_trial, "GPUGMLMPop_dataset_GPU errors: could not free dW_trial");

    //clear the groups
    for(auto gg : Groups) {
        delete gg;
    }
}

template <class FPTYPE>
GPUGMLMPop_dataset_Group_GPU<FPTYPE>::~GPUGMLMPop_dataset_Group_GPU() {
    cudaSafeFreeVector(X, "GPUGMLMPop_dataset_Group_GPU errors: could not free X[dd]");
    cudaSafeFreeVector(XF, "GPUGMLMPop_dataset_Group_GPU errors: could not free iX[dd]");
    cudaSafeFreeVector(iX, "GPUGMLMPop_dataset_Group_GPU errors: could not free iX[dd]");
    cudaSafeFreeVector(X_temp   , "GPUGMLMPop_dataset_Group_GPU errors: could not free X_temp[dd]");
    
    cudaSafeFree(isShared, "GPUGMLMPop_dataset_Group_GPU errors: could not free isShared");
    cudaSafeFree(isSharedIdentity, "GPUGMLMPop_dataset_Group_GPU errors: could not free isSharedIdentity");

    cudaSafeFree(lambda_v, "GPUGMLMPop_dataset_Group_GPU errors: could not free lambda_v");
    cudaSafeFreeVector(lambda_d, "GPUGMLMPop_dataset_Group_GPU errors: could not free lambda_d[dd]");
    cudaSafeFree(   phi_d, "GPUGMLMPop_dataset_Group_GPU errors: could not free phi_d");

    cudaSafeFreeVector(spi_rows, "GPUGMLMPop_dataset_Group_GPU errors: could not free spi_rows");
    cudaSafeFreeVector(spi_cols, "GPUGMLMPop_dataset_Group_GPU errors: could not free spi_cols");
    cudaSafeFreeVector(spi_data, "GPUGMLMPop_dataset_Group_GPU errors: could not free spi_data");
    cudaSafeFreeVector(spi_buffer, "GPUGMLMPop_dataset_Group_GPU errors: could not free spi_buffer");
    //destroy any cusparse handles
    for(int dd = 0; dd < spi_S.size(); dd++) {
        if(spi_S[dd] != NULL) {
            checkCudaErrors(hipsparseDestroySpMat(*spi_S[dd]), "GPUGMLMPop_dataset_Group_GPU errors: CUSPARSE failed to destroy spi_S descr.");
            delete spi_S[dd];
        }
        if(spi_phi_d[dd] != NULL) {
            checkCudaErrors(hipsparseDestroyDnVec(*spi_phi_d[dd]), "GPUGMLMPop_dataset_Group_GPU errors: CUSPARSE failed to destroy spi_phi_d descr.");
        	delete spi_phi_d[dd];
        }
        if(spi_lambda_d[dd] != NULL) {
            checkCudaErrors(hipsparseDestroyDnVec(*spi_lambda_d[dd]), "GPUGMLMPop_dataset_Group_GPU errors: CUSPARSE failed to destroy spi_lambda_d descr.");
            delete spi_lambda_d[dd];
        }
    }
}

//=============================================================================================================================================================
//=============================================================================================================================================================
//=============================================================================================================================================================
/*Kernel for each observation in a sparse run, for a group
 * Builds the dense regressor matrix with local regressors
*  ridx_sa_all must be assigned
*/
template <class FPTYPE>
__global__ void kernel_getGroupX_local_full(GPUData_kernel<FPTYPE> X_temp, const GPUData_kernel<FPTYPE> X,
                                    const GPUData_kernel<unsigned int> ridx_sa_all) {
    //get current observation number
    unsigned int tt_start = blockIdx.y * blockDim.y + threadIdx.y;
    size_t row = blockIdx.x * blockDim.x + threadIdx.x;
    if(row < ridx_sa_all.x) {
        size_t iX_row;
        iX_row = ridx_sa_all[row];

        //for each event 
        for(unsigned int aa = 0; aa < X_temp.z; aa++) {
            //for each regressor (on this thread)
            for(unsigned int tt = tt_start; tt < X_temp.y; tt += blockDim.y * gridDim.y) {
                X_temp(row, tt, aa) = X(iX_row, tt, aa);
            }
        }
    }
}

//functions to multiply the tensor coefficients by the current parameters
template <class FPTYPE>
void GPUGMLMPop_dataset_Group_GPU<FPTYPE>::multiplyCoefficients(const bool isSparseRun, const GPUGMLMPop_parameters_Group_GPU<FPTYPE> * params, const hipStream_t stream, const hipblasHandle_t cublasHandle) {
    checkCudaErrors(set_dim_R(params->dim_R(), stream), "GPUGMLMPop_dataset_Group_GPU errors: could not set dim_R!");
    if(params->dim_R() == 0) {
        return;
    }
    if(params->dim_R() > dim_R_max()) {
        output_stream << "GPUGMLMPop_dataset_Group_GPU errors: dim_R too large for pre-allocated space!";
        msg->callErrMsgTxt(output_stream);
    }

    if(isSparseRun) {
        checkCudaErrors(lambda_v->resize(stream, parent->dim_N_temp, -1, -1), "GPUGMLM_dataset_Group_GPU::multiplyCoefficients errors: could not set size for sparse runs.");
    }
    else {
        checkCudaErrors(lambda_v->resize(stream, parent->lambda->getSize_max(0), -1, -1), "GPUGMLM_dataset_Group_GPU::multiplyCoefficients errors: could not set size for sparse runs.");
    }
    for(int dd = 0; dd < dim_D(); dd++) {
        GPUData<FPTYPE> * X_c = X[dd];
        if(isSparseRun) {
            checkCudaErrors(X_temp[dd]->resize(  stream, parent->dim_N_temp, -1, -1), "GPUGMLMPop_dataset_Group_GPU::multiplyCoefficients errors: could not set size for sparse runs.");
            checkCudaErrors(lambda_d[dd]->resize(stream, parent->dim_N_temp, -1, -1), "GPUGMLMPop_dataset_Group_GPU::multiplyCoefficients errors: could not set size for sparse runs.");
        }
        else {
            checkCudaErrors(lambda_d[dd]->resize(stream, lambda_d[dd]->getSize_max(0),-1,-1), "GPUGMLMPop_dataset_Group_GPU::multiplyCoefficients errors: could not set size for full runs.");
        }
        if((*isSharedIdentity)[dd]) {
            continue;
        }

        if(isSparseRun && !(*isShared)[dd]) {
            // if sparse run and local regressors, build matrix then multiply
            dim3 block_size;
            if(dim_F(dd) > 8) { 
                block_size.y = 8;
            }
            else if(dim_F(dd) >= 4) { 
                block_size.y = 4;
            }
                block_size.y = 1;
            block_size.x = 1024 / block_size.y;
            dim3 grid_size;
            grid_size.x = parent->dim_N_temp / block_size.x + ((parent->dim_N_temp  % block_size.x == 0)? 0:1);
            grid_size.y = 1;
            kernel_getGroupX_local_full<<<grid_size, block_size, 0, stream>>>(X_temp[dd]->device(), X[dd]->device(), 
                                        parent->ridx_sa_all->device());
            checkCudaErrors("GPUGMLMPop_dataset_Group_GPU::multiplyCoefficients errors:  kernel_getGroupX_local_full launch failed");

            X_c = X_temp[dd];
        }

        checkCudaErrors(XF[dd]->resize(stream, X_c->getSize(0)), "GPUGMLMPop_dataset_Group_GPU::multiplyCoefficients errors: could not set matrix size for run.");           
        checkCudaErrors(X_c->GEMM(XF[dd], params->F[dd], cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N), "GPUGMLMPop_dataset_Group_GPU::multiplyCoefficients errors:  X*F -> XF failed");
    }
}

//=============================================================================================================================================================
//=============================================================================================================================================================
//=============================================================================================================================================================

        /*Kernel for each observation, for a group
 * For each component (rr = 0:(dim_R-1)) takes the product of the XT terms into lambda_v, then lambda_v'*V -> lambda 
 * Returns the observation-wise constribution to the rate from this group (lambda) and sets up the dV computation
 *
 * If computing any dT values AND dim_S > 1, needs some dynamic shared memory to make this work on both 1080 and 2080 cards well. Memory size in bytes is dim_S * blockDim.x * sizeof(FPTYPE)
 */
template <class FPTYPE>
__global__ void kernel_getGroupRate(GPUData_kernel<FPTYPE> lambda_v, 
        GPUData_array_kernel<FPTYPE,MAX_DIM_D> lambda_d,
        const GPUData_array_kernel<FPTYPE,MAX_DIM_D> XF,
        const GPUData_array_kernel<FPTYPE,MAX_DIM_D> F,
        const GPUData_array_kernel<int,MAX_DIM_D> iX,
        const GPUData_kernel<bool> isShared,
        const GPUData_kernel<bool> isSharedIdentity,
        const GPUData_kernel<unsigned int> id_a_trialM,
        const GPUData_kernel<FPTYPE> trial_weights, 
        const bool compute_dV, const GPUData_kernel<bool> compute_dF, const bool compute_dT_any,
        const GPUData_kernel<unsigned int> ridx_sa_all, const size_t dim_A) {
    //get current observation number
    extern __shared__ int t_array_0[];
    FPTYPE * t_array = (FPTYPE*)t_array_0; // shared memory for derivative setup

    const size_t row = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int rr_start  = blockIdx.y * blockDim.y + threadIdx.y;

    if(row < lambda_v.x && rr_start < lambda_v.y) {
        size_t iX_row = row; //if full run
        if(ridx_sa_all.y > 0) {
            //if sparse run
            iX_row = ridx_sa_all[row];
        }

        if(trial_weights.y == 0 || trial_weights[id_a_trialM[iX_row]] != 0) { //if trial not censored
            //for each rank
            for(unsigned int rr = rr_start; rr < lambda_v.y; rr += blockDim.y * gridDim.y) { //dim_R = V->Y
                //for each event 
                FPTYPE lv = 0;

                if(compute_dT_any) {
                    for(unsigned int dd = 0 ; dd < XF.N; dd++) {
                        if(compute_dF[dd]) {
                            for(unsigned int aa = 0; aa < lambda_d[dd].z; aa++) {
                                lambda_d[dd](row, rr, aa) = 0;
                            }
                        }
                    }
                }
                for(unsigned int aa = 0; aa < dim_A; aa++) { //over dim_A
                    FPTYPE lv_aa = 1;
                    //for each factor
                    for(unsigned int dd = 0; dd < XF.N; dd++) { //dim_D = XF->N, dim_S = T->N
                        FPTYPE tc = 0;
                        if(isShared[dd]) { //shared regressors
                            int idx_0 = iX[dd](iX_row, aa);
                            if(idx_0 >= 0) {
                                if(isSharedIdentity[dd]) {
                                    if(idx_0 < F[dd].x) {
                                        tc = F[dd](idx_0, rr);
                                    }
                                }
                                else {
                                    if(idx_0 < XF[dd].x) {
                                        tc = XF[dd](idx_0, rr);
                                    }
                                }
                            }
                        }
                        else  { //local regressors
                            tc = XF[dd](row, rr, aa);
                        }

                        lv_aa *= tc;
                        if(compute_dT_any && XF.N > 1) {
                            t_array[dd + threadIdx.x*XF.N] = tc;
                        }
                        else if(tc == 0 ) {
                            break;
                        }
                        
                    } // dd
                    lv += lv_aa;

                    //sets up any dT matrices (doing this here eliminates the need to go back through the XT matrices in a different kernel)
                    //  I do this outside the previous loop because otherwise everything was super slow on the 1080 cards
                    if(compute_dT_any) {
                        for(unsigned int dd = 0 ; dd < XF.N; dd++) {
                            if(compute_dF[dd]) {
                                FPTYPE tt = 1;
                                for(unsigned int dd2 = 0; dd2 < XF.N; dd2++) {
                                    if(dd2 != dd) {
                                        tt *= t_array[dd2 + threadIdx.x*XF.N];
                                    }
                                }
                                lambda_d[dd](row, rr, aa) += tt;
                            }
                        } //dd
                    }
                } // aa
                lambda_v(row, rr) = lv;
            }
        }
    }
}

template <class FPTYPE>
void GPUGMLMPop_dataset_Group_GPU<FPTYPE>::getGroupRate(const bool isSparseRun, const GPUGMLMPop_parameters_Group_GPU<FPTYPE> * params, const GPUGMLMPop_group_computeOptions * opts, const hipStream_t stream, const hipblasHandle_t cublasHandle) { 
    if(params->dim_R() == 0) {
        // set lambda to 0
        FPTYPE * col = parent->lambda->getData_gpu() + groupNum * parent->lambda->getLD_gpu();
        checkCudaErrors(hipMemsetAsync(col, 0, parent->lambda->getSize(0)*sizeof(FPTYPE), stream), "GPUGMLMPop_dataset_Group_GPU::getGroupRate errors: errors setting rate to 0 for dim_R=0 group");
    }
    else {
        dim3 block_size;
        dim3 grid_size;

        if(dim_R() > 8) {
            block_size.y = 4;
        }
        else  if(dim_R() > 4) {
            block_size.y = 2;
        }
        else {
            block_size.y = 1;
        }
        block_size.x = 1024 / block_size.y;
        grid_size.x  = parent->lambda->getSize(0) / block_size.x + ((parent->lambda->getSize(0) % block_size.x == 0)? 0:1);
        grid_size.y  = dim_R() / block_size.y + ((dim_R() % block_size.x == 0)? 0:1);

        bool compute_dT_any = false;
        for(int ss = 0; ss < params->dim_S(); ss++) {
            if(opts->compute_dT[ss]) {
                compute_dT_any = true;
                break;
            }
        }

        size_t size_shared = (compute_dT_any && params->dim_D() > 1) ? (sizeof(FPTYPE) * params->dim_D() * block_size.x) : 0;
        kernel_getGroupRate<<<grid_size, block_size, size_shared, stream>>>( lambda_v->device(),  GPUData<FPTYPE>::assembleKernels(lambda_d), 
                                                                             GPUData<FPTYPE>::assembleKernels(XF),  GPUData<FPTYPE>::assembleKernels(params->F),  GPUData<int>::assembleKernels(iX),
                                                                            isShared->device(), isSharedIdentity->device(),
                                                                            parent->id_a_trialM->device(),
                                                                            params->getTrialWeights()->device(),
                                                                            opts->compute_dV, params->compute_dF->device(), compute_dT_any,
                                                                            parent->ridx_a_all_c->device(), dim_A);
        checkCudaErrors("GPUGMLMPop_dataset_Group_GPU::getGroupRate errors:  kernel_getGroupRate launch failed");

        // multiply lambda_v * V' -> lambda(:, :, groupNum)
        FPTYPE alpha = 1;
        FPTYPE beta  = 0;
        hipblasStatus_t ce =  cublasGEMM(cublasHandle,
                              HIPBLAS_OP_N,
                              HIPBLAS_OP_T,
                              lambda_v->getSize(0), dim_P(), dim_R(),
                              &alpha,
                              lambda_v->getData_gpu(), lambda_v->getLD_gpu(),
                              params->V->getData_gpu(), params->V->getLD_gpu(),
                              &beta,
                              parent->lambda->getData_gpu() + groupNum*parent->lambda->getInc_gpu(), parent->lambda->getLD_gpu());
        checkCudaErrors(ce, "GPUGMLMPop_dataset_Group_GPU::getGroupRate errors:  lambda_v * V' -> lambda(:, :, groupNum) failed");
    }
}

//=============================================================================================================================================================
//=============================================================================================================================================================
//=============================================================================================================================================================

/* Kernel for each entry of the sparse matrix for S*lambda_t -> phi_t (shared regressor compression)
*  sets up the elements of S to be a column of lambda_d
*/
template <class FPTYPE>
__global__ void kernel_set_spi_S( GPUData_kernel<FPTYPE> S,  const GPUData_kernel<FPTYPE> lambda_v,
                               const GPUData_kernel<int> S_idx, const unsigned int col) {
    size_t nn = blockIdx.x * blockDim.x + threadIdx.x;
    if(nn < S.x) {
        S[nn] = lambda_v(S_idx[nn] % lambda_v.x, col);
    }
}

template <class FPTYPE>
__global__ void kernel_PointWiseMultiply_derivativeSetup( GPUData_kernel<FPTYPE> lambda_d,  const GPUData_kernel<FPTYPE> lambda_v) {
    size_t row = blockIdx.x * blockDim.x + threadIdx.x;
    size_t col = blockIdx.y * blockDim.y + threadIdx.y;
    if(row < lambda_d.x && col < lambda_d.y) {
        for(unsigned int zz = 0; zz < lambda_d.z; zz++) {
            lambda_d(row, col, zz) *= lambda_v(row, col);
        }
    }
}

/*Kernel for each observation in a sparse run, for a group
*  ridx_sa_all must be assigned
*/
template <class FPTYPE>
__global__ void kernel_getGroupX_shared_full(GPUData_kernel<FPTYPE> X_temp, const GPUData_kernel<FPTYPE> X,
                                    GPUData_kernel<int> iX,      
                                    GPUData_kernel<unsigned int> ridx_sa_all,
                                    const bool isIdentity)   {
    //get current observation number
    unsigned int tt_start = blockIdx.y * blockDim.y + threadIdx.y;
    size_t row = blockIdx.x * blockDim.x + threadIdx.x;
    if(row < X_temp.x) {
        size_t iX_row;
        iX_row = ridx_sa_all[row];

        //for each regressor (on this thread)
        for(unsigned int tt = tt_start; tt < X.y; tt += blockDim.y * gridDim.y) {
            //for each event 
            for(unsigned int aa = 0; aa < iX.y; aa++) {
                int idx_0 = iX(iX_row, aa);
                if(idx_0 < 0 || idx_0 >= X.x) {
                    X_temp(row, tt, aa) = 0;
                }
                else {
                    if(isIdentity) {
                        X_temp(row, tt, aa) = (idx_0 == tt) ?  1 : 0;
                    }
                    else {
                        X_temp(row, tt, aa) = X(idx_0, tt);
                    }
                }
            }
        }
    }
}

template <class FPTYPE>
void GPUGMLMPop_dataset_Group_GPU<FPTYPE>::computeDerivatives(GPUGMLMPop_results_Group_GPU<FPTYPE> * results, const bool isSparseRun, GPUGMLMPop_parameters_Group_GPU<FPTYPE> * params, const GPUGMLMPop_group_computeOptions * opts, const hipStream_t stream, const hipblasHandle_t cublasHandle, const hipsparseHandle_t cusparseHandle) {
    if(params->dim_R() == 0) {
        return; //nothing to compute
    }

    if(opts->compute_dV) {
        //for each neuron
         checkCudaErrors(parent->dLL->GEMM(results->dV, lambda_v, cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N), "GPUGMLMPop_dataset_Group_GPU::computeDerivatives errors:  dLL'*lambda_v -> dV failed");
    }

    //check if computing any derivatives first
    std::vector<bool> compute_dF;
    compute_dF.assign(dim_D(), false);
    for(int ss = 0; ss < params->dim_S(); ss++) {
        unsigned int dd = (*(params->factor_idx))[ss];
        compute_dF[dd] = compute_dF[dd] || opts->compute_dT[ss];
    }

    // compute lambda_v = dLL * V
    for(int dd = 0; dd < dim_D(); dd++) {
        if(compute_dF[dd]) {
            checkCudaErrors(parent->dLL->GEMM(lambda_v, params->V, cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N), "GPUGMLMPop_dataset_Group_GPU::computeDerivatives errors:  dLL*V -> lambda_v failed");
            break;
        }
    }

    //for each factor
    for(int dd = 0; dd < dim_D(); dd++) {
        if(compute_dF[dd]) {
            // lambda_d init setup in the kernel call in computeRateParts 
            // two steps
            //  lambda_d = lambda_d .* lambda_v
            //  matrix mult of X'*(lambda_d)
                    
            GPUData<FPTYPE> * phi_c;
            GPUData<FPTYPE> * X_c;
            if((*isShared)[dd] && !isSparseRun) { // only do this if doing full run
                //this step is faster with sparse matrices for shared regressors

                //call kernel to setup entries spi to dLL 
                dim3 block_size;
                block_size.x = 1024;
                dim3 grid_size;
                grid_size.x = spi_rows[dd]->size()/ block_size.x + ((spi_rows[dd]->size() % block_size.x == 0)? 0:1);

                FPTYPE alpha = 1;
                FPTYPE beta  = 0;
                for(int rr = 0; rr < params->dim_R(); rr++) {
                    kernel_set_spi_S<<<grid_size, block_size, 0, stream>>>(spi_data[dd]->device(), lambda_v->device(),
                                                         spi_cols[dd]->device(), rr);
                    checkCudaErrors("GPUGMLMPop_dataset_Group_GPU::computeDerivatives errors:  kernel_set_spi_S launch failed");

                    //I found - on a 1080ti at least - doing this series of SpMV ops was typically faster than a single SpMM (annoyingly)
                    hipsparseStatus_t cusparse_stat;
                    cusparse_stat = hipsparseDnVecSetValues(*(spi_lambda_d[dd]), lambda_d[dd]->getData_gpu() + rr*lambda_d[dd]->getLD_gpu());
                    checkCudaErrors(cusparse_stat, "GPUGMLMPop_dataset_Group_GPU errors: hipsparseDnVecSetValues failed for lambda_t.");
                    if((*isSharedIdentity)[dd]) {
                        cusparse_stat = hipsparseDnVecSetValues(*(spi_phi_d[dd]), results->dF[dd]->getData_gpu() + rr*results->dF[dd]->getLD_gpu());
                    }
                    else {
                        cusparse_stat = hipsparseDnVecSetValues(*(spi_phi_d[dd]), phi_d->getData_gpu() + rr*phi_d->getLD_gpu());
                    }
                    checkCudaErrors(cusparse_stat, "GPUGMLMPop_dataset_Group_GPU errors: hipsparseDnVecSetValues failed for phi_d.");
                       
                    cusparse_stat = hipsparseSpMV(cusparseHandle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha,
                                 *(spi_S[dd]),
                                 *(spi_lambda_d[dd]),
                                 &beta,
                                 *(spi_phi_d[dd]),
                                 getCudaType<FPTYPE>(),
                                 HIPSPARSE_SPMV_ALG_DEFAULT,
                                 spi_buffer[dd]->getData_gpu());
                    checkCudaErrors(cusparse_stat, "GPUGMLMPop_dataset_Group_GPU errors: S*lambda->phi_t SpMV failed.");
                }

                X_c   = X[dd];
                phi_c = phi_d;
            }
            else { 
                if((*isShared)[dd]) { 
                    //  if doing sparse run with shared regressor, builds temporary X matrix (local regressors)
                    dim3 block_size;
                    block_size.y = 1;
                    block_size.x = 1024 / block_size.y;
                    dim3 grid_size;
                    grid_size.x = X_temp[dd]->getSize(0)  / block_size.x + ((X_temp[dd]->getSize(0)  % block_size.x == 0)? 0:1);
                    grid_size.y = 1;

                    kernel_getGroupX_shared_full<<<grid_size, block_size, 0, stream>>>(X_temp[dd]->device(), X[dd]->device(), 
                                                    iX[dd]->device(), 
                                                    parent->ridx_sa_all->device(),
                                                    (*isSharedIdentity)[dd]);
                    checkCudaErrors("GPUGMLMPop_dataset_Group_GPU::computeDerivatives errors:  kernel_getGroupX_shared_full launch failed");
                }
                
                // if local regressors
                dim3 block_size;
                if(dim_R() > 8) {
                    block_size.y = 4;
                }
                else  if(dim_R() > 4) {
                    block_size.y = 2;
                }
                else {
                    block_size.y = 1;
                }
                block_size.x = 1024 / block_size.y;
                dim3 grid_size;
                grid_size.x = lambda_v->getSize(0) / block_size.x + ((lambda_v->getSize(0) % block_size.x == 0)? 0:1);
                grid_size.y = dim_R()  / block_size.y + ((dim_R()  % block_size.y == 0)? 0:1);
                        
                kernel_PointWiseMultiply_derivativeSetup<<<grid_size, block_size, 0, stream>>>(lambda_d[dd]->device(), lambda_v->device());
                checkCudaErrors("GPUGMLMPop_dataset_Group_GPU::computeDerivatives errors:  kernel_PointWiseMultiply_derivativeSetup launch failed");

                if(isSparseRun) {
                    // if sparse run
                    X_c     = X_temp[dd];
                    phi_c = lambda_d[dd];
                }
                else {
                    // if local regressors and full run
                    X_c   = X[dd];
                    phi_c = lambda_d[dd];
                }
            }

            checkCudaErrors(phi_c->resize(stream, X_c->getSize(0), results->dF[dd]->getSize(1), X_c->getSize(2)), "GPUGMLMPop_dataset_Group_GPU::computeDerivatives errors: setting size of phi_c failed");

            // matrix mult to get dF (local and shared)
            if((*isShared)[dd] && !isSparseRun && (*isSharedIdentity)[dd]) {
                //nothing needed
            }
            else {
                checkCudaErrors(X_c->GEMM(results->dF[dd], phi_c, cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N), "GPUGMLMPop_dataset_Group_GPU::computeDerivatives errors:   X'*phi -> dF");
            }
            
            // matrix mults to get dT
            if((*(params->N_per_factor))[dd] > 1) {
                for(int ss = 0; ss < params->dim_S(); ss++) {
                    if((*(params->factor_idx))[ss] == dd && opts->compute_dT[ss]) {
                        checkCudaErrors(params->dF_dT[ss]->GEMVs(results->dT[ss], results->dF[dd], cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N), "GPUGMLMPop_dataset_Group_GPU::computeDerivatives errors: dF_dT'*dF -> dT");
                    }
                }
            }
        }
    }
}

//=============================================================================================================================================================
//=============================================================================================================================================================
//=============================================================================================================================================================
//explicitly create classes for single and double precision floating point for library
template class GPUGMLMPop_parameters_Group_GPU<float>;
template class GPUGMLMPop_parameters_Group_GPU<double>;
template class GPUGMLMPop_parameters_GPU<float>;
template class GPUGMLMPop_parameters_GPU<double>;

template class GPUGMLMPop_results_Group_GPU<float>;
template class GPUGMLMPop_results_Group_GPU<double>;
template class GPUGMLMPop_results_GPU<float>;
template class GPUGMLMPop_results_GPU<double>;

template class GPUGMLMPop_dataset_Group_GPU<float>;
template class GPUGMLMPop_dataset_Group_GPU<double>;
template class GPUGMLMPop_dataset_GPU<float>;
template class GPUGMLMPop_dataset_GPU<double>;

};//end namespace