#include "hip/hip_runtime.h"
/*
 * kcGMLMPop_computeBlock.cu
 * Computations for a GMLMPop+derivatives (on one GPU).
 *
 * Package GMLM_dmc for dimensionality reduction of neural data.
 *   
 *  References
 *   Kenneth Latimer & David Freeedman (2021). Low-dimensional encoding of 
 *   decisions in parietal cortex reflects long-term training history.
 *   bioRxiv
 *
 *  Copyright (c) 2021 Kenneth Latimer
 *
 *   This software is distributed under the GNU General Public
 *   License (version 3 or later); please refer to the file
 *   License.txt, included with the software, for details.
 */
#include "kcGMLM_dataStructures.hpp"
#include "kcGMLMPop_computeBlock.hpp"

namespace kCUDA {
    
template <class FPTYPE>
GPUGMLMPop_computeBlock<FPTYPE>::GPUGMLMPop_computeBlock(const GPUGMLM_structure_args<FPTYPE> * GMLMPopstructure, const GPUGMLM_GPU_block_args<FPTYPE> * block, const size_t max_trials_, std::shared_ptr<GPUGL_msg> msg_) {
    this->msg = msg_;
    this->dev  = block->dev_num;
    this->switchToDevice();
    this->checkDeviceComputeCapability();
    dim_J = GMLMPopstructure->Groups.size();

    size_t dim_M = block->trials.size();
    if(dim_M == 0) {
        this->output_stream << "GPUGMLMPop_computeBlock errors: no trials in block!";
        this->msg->callErrMsgTxt(this->output_stream);
    }   

    //setup the streams
    this->checkCudaErrors(hipStreamCreate(&(stream)), "GPUGMLMPop_computeBlock errors: failed initializing stream!");
    stream_Groups.resize(dim_J);
    for(unsigned int jj = 0; jj < dim_J; jj++) {
        this->checkCudaErrors(hipStreamCreate(&(stream_Groups[jj])), "GPUGMLMPop_computeBlock errors: failed initializing group streams!");
    }

    //setup cublas handles
    hipblasMath_t mathMode = HIPBLAS_DEFAULT_MATH;
    #if __CUDA_ARCH__ >= 700
        mathMode = HIPBLAS_TF32_TENSOR_OP_MATH;
    #endif
    
    this->checkCudaErrors(hipblasCreate(&(cublasHandle)), "GPUGMLMPop_computeBlock errors: CUBLAS initialization failed.");
    this->checkCudaErrors(hipblasSetStream(cublasHandle, stream), "GPUGMLMPop_computeBlock errors: set cublas stream failed.");
    this->checkCudaErrors(hipblasSetMathMode(cublasHandle, mathMode), "GPUGMLMPop_computeBlock errors: set cublas math mode failed.");
    this->checkCudaErrors(hipblasSetPointerMode(cublasHandle, HIPBLAS_POINTER_MODE_HOST), "GPUGMLMPop_computeBlock errors: set cublas pointer mode failed.");

    cublasWorkspace = NULL;
    /*size_t cublasWorkspace_size_0 = 1024 * 1024 * 0;	// if greater than 0, sets special workspace size (doesn't seem to help the current computations)	
    if(cublasWorkspace_size_0 > 0) {
        this->checkCudaErrors(hipMallocPitch(reinterpret_cast<void**>(&(cublasWorkspace)), &cublasWorkspace_size, cublasWorkspace_size_0, 1), "GPUGMLMPop_computeBlock errors: allocating cublas workspace failed.");
        this->checkCudaErrors(hipblasSetWorkspace(cublasHandle, cublasWorkspace, cublasWorkspace_size), "GPUGMLMPop_computeBlock errors: setting CUBLAS workspace failed.");
    }*/


    cublasHandle_Groups.resize(dim_J);
    cublasWorkspaces.assign(dim_J, NULL);
    cublasWorkspaces_size.assign(dim_J, cublasWorkspace_size);
    for(unsigned int jj = 0; jj < dim_J; jj++) {
        this->checkCudaErrors(hipblasCreate(&(cublasHandle_Groups[jj])), "GPUGMLMPop_computeBlock errors: CUBLAS groups initialization failed.");
        this->checkCudaErrors(hipblasSetMathMode(cublasHandle_Groups[jj], mathMode), "GPUGMLMPop_computeBlock errors: set cublas group math mode failed.");
        this->checkCudaErrors(hipblasSetPointerMode(cublasHandle_Groups[jj], HIPBLAS_POINTER_MODE_HOST), "GPUGMLMPop_computeBlock errors: set cublas groups pointer mode failed.");
        this->checkCudaErrors(hipblasSetStream(cublasHandle_Groups[jj], stream_Groups[jj]), "GPUGMLMPop_computeBlock errors: set cublas groups stream failed.");

     /*   if(cublasWorkspaces_size[jj] > 0) {
            this->checkCudaErrors(hipMallocPitch(reinterpret_cast<void**>(&(cublasWorkspaces[jj])), &cublasWorkspaces_size[jj], cublasWorkspace_size_0, 1), "GPUGMLMPop_computeBlock errors: allocating group cublas workspace failed.");
            this->checkCudaErrors(hipblasSetWorkspace(cublasHandle_Groups[jj], cublasWorkspaces[jj], cublasWorkspaces_size[jj]), "GPUGMLMPop_computeBlock errors: setting group CUBLAS workspace failed.");
        }*/
    }

    //setup cusparse handle
    cusparseHandle_Groups.resize(dim_J);
    for(unsigned int jj = 0; jj < dim_J; jj++) {
        this->checkCudaErrors(hipsparseCreate(       &(cusparseHandle_Groups[jj])), "GPUGMLMPop_computeBlock errors: cusparse groups initialization failed.");
        this->checkCudaErrors(hipsparseSetPointerMode(cusparseHandle_Groups[jj], HIPSPARSE_POINTER_MODE_HOST), "GPUGMLMPop_computeBlock errors: set cusparse groups pointer mode failed.");
        this->checkCudaErrors(hipsparseSetStream(      cusparseHandle_Groups[jj], stream_Groups[jj]), "GPUGMLMPop_computeBlock errors: set cusparse groups stream failed.");
    }

    //setup the parameter structure
    params = new GPUGMLM_parameters_GPU<FPTYPE>(GMLMPopstructure, dim_M, this->dev, this->msg);
    //params = NULL;

    //setup the results structure
    results = new GPUGMLM_results_GPU<FPTYPE>(GMLMPopstructure, max_trials_, this->dev, this->msg);
    //results = NULL;
            
    //setup the dataset structure
    dataset = new GPUGMLMPop_dataset_GPU<FPTYPE>(GMLMPopstructure, block, max_trials_, stream, cusparseHandle_Groups, this->msg);
    //dataset = NULL;
    this->checkCudaErrors(hipEventCreate(&LL_event), "GPUGMLMPop_computeBlock errors: could not create LL event!");
}

template <class FPTYPE>
GPUGMLMPop_computeBlock<FPTYPE>::~GPUGMLMPop_computeBlock() {
    this->switchToDevice();
    
    delete results;
    delete params;
    delete dataset;

    this->checkCudaErrors(hipEventDestroy(LL_event), "GPUGMLMPop_computeBlock errors: could not clear LL event!");

    //destroy cublas handles
    this->checkCudaErrors(hipblasDestroy(cublasHandle), "GPUGMLMPop_computeBlock errors: failed to destroy cublas handle." );
    for(auto jj : cublasHandle_Groups) {
        this->checkCudaErrors(hipblasDestroy(jj), "GPUGMLMPop_computeBlock errors: failed to destroy group cublas handles." );
    }
    for(auto jj : cusparseHandle_Groups) {
        this->checkCudaErrors(hipsparseDestroy(jj), "GPUGMLMPop_computeBlock errors: failed to destroy group cusparse handles." );
    }
       
    this->cudaSafeFreePtr(cublasWorkspace, "GPUGMLMPop_computeBlock errors: failed to destroy cublas workspace." );
    this->cudaSafeFreePtrVector(cublasWorkspaces, "GPUGMLMPop_computeBlock errors: failed to destroy cublas group workspaces." );
    //destroy streams
    this->checkCudaErrors(hipStreamDestroy(stream), "GPUGMLMPop_computeBlock errors: failed destroying stream!");
    for(auto jj : stream_Groups) {
        this->checkCudaErrors(hipStreamDestroy(jj), "GPUGMLMPop_computeBlock errors: failed to destroy group streams." );
    }  
}

template <class FPTYPE>
bool GPUGMLMPop_computeBlock<FPTYPE>::loadParams(const GPUGMLM_params<FPTYPE> * params_host, const GPUGMLM_computeOptions<FPTYPE> * opts) { 
    this->switchToDevice();
    params->copyToGPU(params_host, dataset, stream, stream_Groups, opts);
    for(unsigned int jj = 0; jj < params->dim_J(); jj++) {
        this->checkCudaErrors(results->set_dim_R(jj, params->dim_R(jj), stream), "GPUGMLMPop_computeBlock::loadParams errors: could not set results dim_R");
    }
    bool isSparseRun = dataset->isSparseRun(params);
    if(params->getNumberOfNonzeroWeights() > 0) { //make sure there's something to compute
        results_set = true;

        //for each group, multiply coefficients by X*T -> XT
        for(unsigned int jj = 0; jj < dim_J && jj < dataset->dim_J(); jj++) {
            dataset->Groups[jj]->multiplyCoefficients(isSparseRun, opts->update_weights, params->Groups[jj], stream_Groups[jj], cublasHandle_Groups[jj], params->paramsLoaded_event);
        }
    }
    else {
        results_set = false;
    }
    return isSparseRun;
}
        
template <class FPTYPE>
void GPUGMLMPop_computeBlock<FPTYPE>::computeRateParts(const GPUGMLM_computeOptions<FPTYPE> * opts, const bool isSparseRun) {
    if(params->getNumberOfNonzeroWeights() == 0) { //nothing to compute
        return;
    }
    this->switchToDevice();

    //for each group
    for(unsigned int jj = 0; jj < dataset->dim_J(); jj++ ) {
        dataset->Groups[jj]->getGroupRate(isSparseRun,  params->Groups[jj], opts->Groups[jj], stream_Groups[jj], cublasHandle_Groups[jj]);
    }
}


/*Kernel for each observation
 *  for sparse runs with compute_dB, saves out the partial X_lin into X_lin_temp
 */        
template <class FPTYPE>
__global__ void kernel_setup_X_lin_temp(
        const GPUData_kernel<FPTYPE> X_lin ,
        const GPUData_kernel<unsigned int> id_a_trialM,
        const GPUData_kernel<FPTYPE> trial_weights,
        const GPUData_kernel<unsigned int> ridx_sa_all,
        GPUData_kernel<FPTYPE> X_lin_temp) {
    //current observation index

    for(size_t row_0 = blockIdx.x * blockDim.x; row_0 < X_lin_temp.x; row_0 += blockDim.x * gridDim.x) {
        size_t row = row_0 + threadIdx.x;
        size_t Xlin_row = row; //if full run
        if(ridx_sa_all.y > 0 && row < ridx_sa_all.x) {
            //if sparse run
            Xlin_row = ridx_sa_all[row];
        }
        FPTYPE tw_c = 1;
        if(row < X_lin_temp.x && trial_weights.y == 1) {
            unsigned int tr_idx = id_a_trialM[Xlin_row];
            if(trial_weights.x > tr_idx) {
                tw_c = trial_weights[tr_idx];
            }
        }
        __syncthreads();
        for(unsigned int pp_0 = blockIdx.y * blockDim.y; pp_0 < X_lin_temp.z; pp_0 += blockDim.y * gridDim.y) {
            unsigned int pp = pp_0 + threadIdx.y;

            if(X_lin_temp.z > 1 && row < X_lin_temp.x && trial_weights.y > 1 && pp < trial_weights.y) {
                unsigned int tr_idx = id_a_trialM[Xlin_row];
                if(trial_weights.x > tr_idx) {
                    tw_c = trial_weights(tr_idx, pp);
                }
            }

            bool elementIncluded = row < X_lin_temp.x && pp < X_lin_temp.z && tw_c != 0;

            __syncwarp();

            for(int bb = 0; bb < X_lin.y; bb++) {
                if(elementIncluded) {
                    if(ridx_sa_all.y > 0 && (pp == 0 || X_lin_temp.z > 1)) { // for dB when doing sparse run
                        X_lin_temp(row, bb, pp) = X_lin(Xlin_row, bb, pp);
                    }
                }
                __syncwarp();
            }
            __syncthreads();
        }
    }
}

/*Kernel for each observation
 * Summarizes the contributions from each tensor group (lambda), linear term (X_lin,B), baseline rate (w,log_dt)
 * Returns the  observation-wise log like (LL - no normalizing constant) and it's derivative portion (dLL)
 *
 */        
template <class FPTYPE>
__global__ void kernel_getObs_LL_pop(GPUData_kernel<FPTYPE> LL, GPUData_kernel<FPTYPE> dLL,
        const GPUData_kernel<FPTYPE> Y,
        const GPUData_kernel<FPTYPE> lambda,
        bool addW,
        const GPUData_kernel<FPTYPE> W, 
        const FPTYPE log_dt, const FPTYPE dt,
        const GPUData_kernel<unsigned int> id_a_trialM,
        const GPUData_kernel<FPTYPE> trial_weights,
        const GPUData_kernel<unsigned int> ridx_sa_all,
        const logLikeType logLikeSettings, const GPUData_kernel<FPTYPE> logLikeParams) {
    //current observation index

    for(size_t row_0 = blockIdx.x * blockDim.x; row_0 < LL.x; row_0 += blockDim.x * gridDim.x) {
        size_t row = row_0 + threadIdx.x;
        size_t Xlin_row = row; //if full run
        if(ridx_sa_all.y > 0 && row < ridx_sa_all.x) {
            //if sparse run
            Xlin_row = ridx_sa_all[row];
        }
        FPTYPE tw_c = 1;
        if(row < LL.x && trial_weights.y == 1) {
            unsigned int tr_idx = id_a_trialM[Xlin_row];
            if(trial_weights.x > tr_idx) {
                tw_c = trial_weights[tr_idx];
            }
        }
        __syncthreads();
        for(unsigned int pp_0 = blockIdx.y * blockDim.y; pp_0 < LL.y; pp_0 += blockDim.y * gridDim.y) {
            unsigned int pp = pp_0 + threadIdx.y;
            if(row < LL.x && trial_weights.y > 1 && pp < trial_weights.y) {
                unsigned int tr_idx = id_a_trialM[Xlin_row];
                if(trial_weights.x > tr_idx) {
                    tw_c = trial_weights(tr_idx, pp);
                }
            }

            bool elementIncluded = row < LL.x && pp < LL.y && tw_c != 0;
        
            FPTYPE  LL_c = 0;  
            FPTYPE dLL_c = 0;
            FPTYPE log_rate = 0; 
            FPTYPE Y_c  = 0;   

            if(elementIncluded) {
                Y_c = Y(Xlin_row, pp);
                if(!addW) {
                    log_rate = W[pp];
                }
                else {
                    log_rate = W[pp] + LL(row, pp);
                }
            }
            __syncwarp();

            
            for(int jj = 0; jj < lambda.z; jj++) {
                if(elementIncluded) {
                    log_rate += lambda(row, pp, jj);
                }
                __syncwarp();
            }
            __syncthreads();


            if(elementIncluded) {
                if(logLikeSettings == ll_poissExp) {
                    if(Y_c >= 0) { // negatives get censored by Poisson LL
                        Y_c = floor(Y_c);
                        log_rate += log_dt;
                        FPTYPE rate = safeExp(log_rate);
                        LL_c = (-rate + Y_c * log_rate);
                        dLL_c = (-rate + Y_c);
                    }
                }
                else if(logLikeSettings == ll_poissSoftRec) {
                    if(Y_c >= 0) { // negatives get censored by Poisson LL
                        Y_c = floor(Y_c);
                        FPTYPE rate;
                        FPTYPE drate;
                        FPTYPE drate_rate;
                        if(log_rate > 30) {
                            rate  = log_rate ; // in this model, log_dt is actually just dt
                            drate = 1;
                            drate_rate = 1.0/log_rate;
                        }
                        else {
                            log_rate = log_rate < -30 ? -30 : log_rate; // to be safe with the log
                            rate  = log1p(safeExp(log_rate));
                            drate = (1.0 + safeExp(-log_rate));
                            drate_rate = 1.0/(drate * rate);
                            drate = 1.0/drate;
                        }
                        LL_c  = (-rate*dt + Y_c *(log(rate) + log_dt));
                        dLL_c = (-drate*dt + Y_c * drate_rate);
                    }
                }
                else if(logLikeSettings == ll_sqErr) {
                    FPTYPE eY_c = log_rate - Y_c;
                    LL_c = -0.5*(eY_c*eY_c);
                    dLL_c = -eY_c;
                }
                else if(logLikeSettings == ll_truncatedPoissExp) {
                    if(Y_c >= 1) { 
                        log_rate += log_dt;
                        if(log_rate > -30) {
                            FPTYPE rate = safeExp(log_rate);
                            LL_c = log(1.0 - safeExp(-rate));
                            dLL_c = rate/safeExpm1(rate);
                        }
                        else { // more numerically save approximation in an extreme case
                            LL_c = log_rate;
                            dLL_c = 1;
                        }
                    }
                    else if(Y_c == 0) {
                        FPTYPE rate = safeExp(log_rate + log_dt);
                        LL_c = -rate;
                        dLL_c = -rate;
                    }
                    // negatives get censored by Poisson LL
                }
                else if(logLikeSettings == ll_poissExpRefractory) {
                    // ll_poissExpRefractory uses the correction from Citi, L., Ba, D., Brown, E. N., & Barbieri, R. (2014). Likelihood methods for point processes with refractoriness. Neural computation, 26(2), 237-263.
                    if(Y_c >= 0) { // negatives get censored by Poisson LL
                        Y_c = floor(Y_c);
                        log_rate += log_dt;
                        FPTYPE rate = safeExp(log_rate);
                        LL_c = (-(1-Y_c/2)*rate + Y_c * log_rate);
                        dLL_c = (-(1-Y_c/2)*rate + Y_c);
                    }
                }

                LL(row, pp) =  LL_c*tw_c;
                dLL(row, pp) = dLL_c*tw_c;
            }
            else if(row < LL.x && pp < LL.y) {
                LL(row, pp) = 0;
                dLL(row, pp) = 0;
            }
            __syncthreads();
        }
    }
}

/* Kernel for each trial
*  Sums up the trial log likelihoods (results->trialLL)
*   also sets up some derivative computations (dataset->dW_trial, dataset->dB_trial)*/
template <class FPTYPE>
__global__ void kernel_sum_trialLL_pop(GPUData_kernel<FPTYPE> trialLL, GPUData_kernel<FPTYPE> dW_trial, 
                                 const GPUData_kernel<unsigned int> trial_included, 
                                 const GPUData_kernel<FPTYPE> LL, const GPUData_kernel<FPTYPE> dLL, 
                                 const bool compute_trialLL, const bool compute_dW, 
                                 const GPUData_kernel<size_t> dim_N,
                                 const GPUData_kernel<unsigned int> ridx_t_all,
                                 const GPUData_kernel<unsigned int> id_t_trial,
                                 const GPUData_kernel<FPTYPE> trial_weights,
                                 const GPUData_kernel<FPTYPE> normalizingConstants) {
    size_t tr = blockIdx.x * blockDim.x + threadIdx.x;
    size_t pp = blockIdx.y * blockDim.y + threadIdx.y;
    size_t mm = dim_N.x; //default is invalid value - will just skip
    if(trial_included.y > 0) { //if is sparse run
        if(tr < trial_included.x) {
            mm = trial_included[tr];
        }
    }
    else {
        mm = tr;
    }

    if(mm < dim_N.x && pp < trialLL.y) { // if valid trial
        FPTYPE tw_c = 1;
        if(trial_weights.y == 1 && trial_weights.x > mm) {
            tw_c = trial_weights[mm];
        }
        else if(trial_weights.y > 1 && trial_weights.x > mm) {
            tw_c = trial_weights(mm, pp);
        }

        if(tw_c != 0) {  
            unsigned int row = ridx_t_all[tr];  // this uses 'tr' so that it works for sparse runs

            //sum up LL
            FPTYPE ll_total = normalizingConstants(mm, pp) * tw_c;
            FPTYPE dll_total = 0;
            for(int tt = 0; tt < dim_N[mm]; tt++) {
                if(compute_trialLL) {
                    ll_total  += LL(row + tt, pp);
                }
                if(compute_dW) {
                    dll_total  += dLL(row + tt, pp);
                }
            }
            if(compute_trialLL) {
                trialLL(id_t_trial[mm], pp) = ll_total;
            }
            if(compute_dW) {
                dW_trial(mm, pp) = dll_total;
            }
        }
        //no need to compute sum; set results to 0
        else {
            if(compute_trialLL) {
                trialLL(id_t_trial[mm], pp) = 0;
            }
            if(compute_dW) {
                dW_trial(mm, pp) = 0;
            }
        }
    }
}

template <class FPTYPE>
void GPUGMLMPop_computeBlock<FPTYPE>::computeLogLike(const GPUGMLM_computeOptions<FPTYPE> * opts, const bool isSparseRun) {
    if(params->getNumberOfNonzeroWeights() == 0) { //nothing to compute
        return;
    }

    this->switchToDevice();
         //launch kernel to sum lambda, X_lin*B -> LL, dLL (launch over all observations)
         //launch kernel to sum lambda for each trial

         
    GPUData<FPTYPE> * X_lin_c = isSparseRun ?  dataset->X_lin_temp : dataset->X_lin;

    if(isSparseRun && opts->update_weights) {
        dataset->X_lin_temp->resize(stream, dataset->LL->getSize(0));

        dim3 block_size;
        block_size.x = 1024/block_size.y;
        dim3 grid_size;
        size_t max_blocks_needed  = dataset->LL->getSize(0) / block_size.x + ( (dataset->LL->getSize(0) % block_size.x == 0) ? 0 : 1);
        size_t blocks_to_use = 1024;
        grid_size.x  = min(max_blocks_needed, blocks_to_use);
        grid_size.y = dataset->X_lin_temp->getSize(2) / block_size.y + ( (dataset->X_lin_temp->getSize(2) % block_size.y == 0) ? 0 : 1);
        kernel_setup_X_lin_temp<<<grid_size, block_size, 0, stream>>>( dataset->X_lin->device(),
                dataset->id_a_trialM->device(),
                params->trial_weights->device(),
                dataset->ridx_a_all_c->device(),
                dataset->X_lin_temp->device());
    }

    //X_lin*B -> LL
    if(dataset->dim_B() > 0) {
        hipblasStatus_t ce;
        if(X_lin_c->getInc_gpu() == 0) {
            ce = X_lin_c->GEMM(dataset->LL, params->B, cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N);
        }       
        else {
            ce = X_lin_c->GEMVs(dataset->LL, params->B, cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N);
        }
        this->checkCudaErrors(ce, "GPUGMLMPop_computeBlock::computeLogLike errors:  X_lin * B launch failed");
    }
    
    //LL + W + log_dt + sum(lambda) -> LL for each neuron
    this->checkCudaErrors(dataset->waitForGroups_LL(stream), "GPUGMLM_computeBlock::computeLogLike errors:  waitForGroups_LL failed");

    dim3 block_size;
    if(params->dim_P() > 8) {
    	block_size.y = 4;
    }
    else if(params->dim_P() > 4) {
    	block_size.y = 2;
    }
    else {
    	block_size.y = 1;
    }
    
        
    block_size.x = 1024/block_size.y;
    dim3 grid_size;
    size_t max_blocks_needed  = dataset->LL->getSize(0) / block_size.x + ( (dataset->LL->getSize(0) % block_size.x == 0) ? 0 : 1);
    size_t blocks_to_use = 1024;
    grid_size.x  = min(max_blocks_needed, blocks_to_use);
    grid_size.y = dataset->dim_P() / block_size.y + ( (dataset->dim_P() % block_size.y == 0) ? 0 : 1);

    /*this->output_stream << " grid_size.y  = " << grid_size.y  << ", " << " block_size.y  = " << block_size.y << "\n";
    this->output_stream << " grid_size.x  = " << grid_size.x  << ", " << " block_size.x  = " << block_size.x << "\n";
    this->output_stream << " dim_P  = " << dataset->dim_P() << ", " << " dataset->LL->getSize(0) = " << dataset->LL->getSize(0) << "\n";
    this->msg->printMsgTxt(this->output_stream);
    this->checkCudaErrors("PRE GPUGMLMPop_computeBlock::computeLogLike errors:  kernel_getObs_LL launch failed");*/

    kernel_getObs_LL_pop<<<grid_size, block_size, 0, stream>>>(dataset->LL->device(), dataset->dLL->device(),
                  dataset->Y->device(),
                  dataset->lambda->device(),
                  dataset->dim_B() > 0,
                   params->W->device(), dataset->log_dt, dataset->dt,
                  dataset->id_a_trialM->device(),
                  params->trial_weights->device(),
                  dataset->ridx_a_all_c->device(),
                   params->logLikeSettings, params->logLikeParams->device());
                   
    this->checkCudaErrors("GPUGMLMPop_computeBlock::computeLogLike errors:  kernel_getObs_LL launch failed");
    this->checkCudaErrors(hipEventRecord(LL_event, stream), "GPUGMLMPop_computeBlock::computeLogLike errors: could not add LL event to stream!");

    //sum up the LL for each trial (and dLL to setup for dW, dB)
    if(opts->compute_trialLL || opts->compute_dW) {
        //same block size

        size_t dim_M_c = params->getNumberOfNonzeroWeights();
        grid_size.x = params->getNumberOfNonzeroWeights()  / block_size.x + ( (params->getNumberOfNonzeroWeights()  % block_size.x == 0) ? 0 : 1);
        grid_size.y = dataset->dim_P() / block_size.y + ( (dataset->dim_P() % block_size.y == 0) ? 0 : 1);

        //this->output_stream << " grid_size.y  = " << grid_size.y  << ", " << " block_size.y  = " << block_size.y << "\n";
        //this->msg->printMsgTxt(output_stream);
        kernel_sum_trialLL_pop<<<grid_size, block_size, 0, stream>>>(results->trialLL->device(), dataset->dW_trial->device(),
                                                                 params->trial_included->device(), 
                                                                 dataset->LL->device(), dataset->dLL->device(), 
                                                                 opts->compute_trialLL, opts->compute_dW, 
                                                                 dataset->dim_N->device(),
                                                                 dataset->ridx_t_all_c->device(),
                                                                 dataset->id_t_trial->device(),
                                                                 params->trial_weights->device(),
                                                                 dataset->normalizingConstants_trial->device());
        this->checkCudaErrors("GPUGMLMPop_computeBlock::computeLogLike errors:  kernel_sum_trialLL launch failed");
    }
}

/* Kernel for each neuron
*  Sums up the trial dW (results->dW)
*   also sets up some derivative computations (dataset->dW_trial, dataset->dB_trial)
*/
template <class FPTYPE>
__global__ void kernel_sum_dW_pop( GPUData_kernel<FPTYPE> dW,  const GPUData_kernel<FPTYPE> dW_trial, const GPUData_kernel<FPTYPE> trial_weights) {
    size_t pp = blockIdx.x * blockDim.x + threadIdx.x;
    if(pp < dW.x) {
        FPTYPE dW_sum = 0;
        for(int tr = 0; tr < dW_trial.x; tr++) {
            if(trial_weights.y == 0 || (trial_weights.y == 1 && trial_weights[tr] != 0) || (trial_weights.y > 1 && trial_weights(tr,pp) != 0)) {
                dW_sum += dW_trial(tr,pp);
            }
        }
        dW[pp] = dW_sum;
    }
}


template <class FPTYPE>
void GPUGMLMPop_computeBlock<FPTYPE>::computeDerivatives(const GPUGMLM_computeOptions<FPTYPE> * opts, const bool isSparseRun) {
    if(params->getNumberOfNonzeroWeights() == 0) { //nothing to compute
        return;
    }
    this->switchToDevice();
         //launch kernel to sum dLL -> dW, dB for each trial?
         //         or kernel to sum up dLL->dW and GEMV for dB?
         
    //for each Group
    for(unsigned int jj = 0; jj < dim_J; jj++) {
        dataset->Groups[jj]->computeDerivatives(results->Groups[jj], isSparseRun, opts->update_weights, params->Groups[jj], opts->Groups[jj], stream_Groups[jj], cublasHandle_Groups[jj], cusparseHandle_Groups[jj], LL_event);
    }   
    
    if(opts->compute_dW) {
        dim3 block_size;
        block_size.x = min(dataset->dim_P(), static_cast<size_t>(1024));
        dim3 grid_size;
        grid_size.x = dataset->dim_P() / block_size.x + ((dataset->dim_P() % block_size.x == 0)? 0:1);

      /*  this->output_stream << results->dW->getDevice() << "  " << dataset->dW_trial->getDevice() << "  " << params->trial_weights->getDevice() << "\n";
        this->msg->printMsgTxt(output_stream);*/


        kernel_sum_dW_pop<<<grid_size, block_size, 0, stream>>>(results->dW->device(), dataset->dW_trial->device(), params->trial_weights->device());
        this->checkCudaErrors("GPUGMLMPop_computeBlock::computeDerivatives errors:  kernel_sum_dW launch failed");
    }

    if(opts->compute_dB && dataset->dim_B() > 0) {
        GPUData<FPTYPE> * X_lin_c = isSparseRun ?  dataset->X_lin_temp : dataset->X_lin;


        hipblasStatus_t ce;
        if(X_lin_c->getSize(2) == 1) { //if one shared X_lin term
            ce = X_lin_c->GEMM(results->dB,  dataset->dLL, cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N);
        }
        else { // X_lin terms for each neuron
            ce = X_lin_c->GEMVs(results->dB, dataset->dLL, cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N);
        }
        this->checkCudaErrors(ce, "GPUGMLMPop_computeBlock::computeDerivatives errors:  X_lin'*dLL -> dB failed");
    }      
}


//============================================================================================================================
//Dataset class
        
//Constructor takes in all the group data and GMLMPop setup
template <class FPTYPE>
GPUGMLMPop_dataset_GPU<FPTYPE>::GPUGMLMPop_dataset_GPU(const GPUGMLM_structure_args<FPTYPE> * GMLMPopstructure, const GPUGMLM_GPU_block_args <FPTYPE> * block, const size_t max_trials_, const hipStream_t stream, const std::vector<hipsparseHandle_t> & cusparseHandle_Groups, std::shared_ptr<GPUGL_msg> msg_) {
    this->dev  = block->dev_num;
    this->msg = msg_;
    this->switchToDevice();
    hipError_t ce;

    dt = GMLMPopstructure->binSize;
    log_dt = log(GMLMPopstructure->binSize);

    Groups.assign(GMLMPopstructure->Groups.size(), NULL); //sets up dim_J()
    dim_N = new GPUData<size_t>(ce, GPUData_HOST_STANDARD, stream, block->trials.size()); //sets up dim_M()
    this->checkCudaErrors(ce, "GPUGMLMPop_dataset_GPU errors: could not allocate dim_N on device!");
            
    // number of trials
    isInDataset_trial.assign( max_trials_, false); //if each trial is in this block
    if(dim_M() == 0) {
        this->output_stream << "GPUGMLMPop_dataset_GPU errors: no trials given to GPU block!";
        this->msg->callErrMsgTxt(output_stream);
    }

    max_trials_for_sparse_run = min(dim_M()/2, static_cast<size_t>(block->max_trials_for_sparse_run));

    // setup up the order that trials go to the GPU
    //   in blocks ordered by neurons     

    size_t dim_N_total_c = 0;
    dim_N_temp = 0;
    max_trial_length = 1;

    ridx_t_all = new GPUData<unsigned int>(ce, GPUData_HOST_STANDARD, stream, dim_M());
    this->checkCudaErrors(ce, "GPUGMLMPop_dataset_GPU errors: could not allocate ridx_t_all on device!");
    id_t_trial = new GPUData<unsigned int>(ce, GPUData_HOST_STANDARD, stream, dim_M());
    this->checkCudaErrors(ce, "GPUGMLMPop_dataset_GPU errors: could not allocate id_t_trial on device!");
    normalizingConstants_trial = new GPUData<FPTYPE>(ce, GPUData_HOST_STANDARD, stream, dim_M(), GMLMPopstructure->dim_P);
    this->checkCudaErrors(ce, "GPUGMLMPop_dataset_GPU errors: could not allocate normalizingConstants_trial on device!");

    size_t X_lin_depth;
    for(unsigned int mm = 0; mm < dim_M(); mm++) {
        if(mm == 0) {
            X_lin_depth = block->trials[mm]->X_lin->getSize(2);
        }

        if(block->trials[mm]->X_lin->getSize(2) != X_lin_depth || (X_lin_depth > 1 && X_lin_depth != GMLMPopstructure->dim_P)) {
            this->output_stream << "GPUGMLMPop_dataset_GPU errors: invalid size of X_lin: depth must be 0-1 or dim_P!";
            this->msg->callErrMsgTxt(output_stream);
        }
        
        //save trial indices
        (*ridx_t_all)[mm] = dim_N_total_c;

        // get trial length
        (*dim_N)[mm] = block->trials[mm]->dim_N(msg);
        if((*dim_N)[mm] == 0) {
            this->output_stream << "GPUGMLMPop_dataset_GPU errors: trials cannot be empty!";
            this->msg->callErrMsgTxt(output_stream);
        }
        dim_N_total_c += (*dim_N)[mm]; // add length to total 

        max_trial_length = max(max_trial_length, (*dim_N)[mm]); //update max trial length

        //save trial and neuron number
        (*id_t_trial)[mm] = block->trials[mm]->trial_idx;
        if(isInDataset_trial[block->trials[mm]->trial_idx]) { //trial index already found
            this->output_stream << "GPUGMLMPop_dataset_GPU errors: trial indices must be unique!";
            this->msg->callErrMsgTxt(output_stream);
        }

        isInDataset_trial[block->trials[mm]->trial_idx] = true;

        for(unsigned int pp = 0; pp < GMLMPopstructure->dim_P; pp++) {
            FPTYPE nc = 0; // normalizing constant
            for(unsigned int nn = 0; nn < (*dim_N)[mm]; nn++) {
                if(GMLMPopstructure->logLikeSettings == ll_poissExp || GMLMPopstructure->logLikeSettings == ll_poissSoftRec) {
                    FPTYPE Y_c = (*(block->trials[mm]->Y))(nn, pp);
                    nc += (Y_c >= 0) ? -lgamma(floor(Y_c) + 1.0) : 0;
                }
            }
            (*normalizingConstants_trial)(mm, pp) = nc;
        }
    }
    id_a_trialM = new GPUData<unsigned int>(ce, GPUData_HOST_STANDARD, stream, dim_N_total_c);
    this->checkCudaErrors(ce, "GPUGMLMPop_dataset_GPU errors: could not allocate id_a_trialM on device!");

    size_t N_total_ctr = 0;
    for(unsigned int mm = 0; mm < dim_M(); mm++) {
        for(unsigned int nn = 0; nn < (*dim_N)[mm]; nn++) {
            (*id_a_trialM)[N_total_ctr + nn] = mm;
        }
        N_total_ctr += (*dim_N)[mm];
    }

    //allocate space on GPU for data and copy any local values to GPU
        //spike counts
    Y = new GPUData<FPTYPE>(ce, GPUData_HOST_NONE, stream, dim_N_total_c, GMLMPopstructure->dim_P);
    this->checkCudaErrors(ce, "GPUGMLMPop_dataset_GPU errors: could not allocate Y on device!");
    
        //linear term (divded up into per-neuron blocks)
    X_lin = new GPUData<FPTYPE>(ce, GPUData_HOST_NONE, stream, dim_N_total_c, GMLMPopstructure->dim_B, X_lin_depth);
    this->checkCudaErrors(ce, "GPUGMLMPop_dataset_GPU errors: could not allocate X_lin on device!");
        
        //copy each trial to GPU
    for(unsigned int mm = 0; mm < dim_M(); mm++) {
        // spike counts
        hipPos copyOffset = make_hipPos((*ridx_t_all)[mm], 0, 0);
        this->checkCudaErrors(Y->copyTo(stream, block->trials[mm]->Y, true, copyOffset), "GPUGMLMPop_dataset_GPU errors: could not copy Y to device!");
                
        // linear term
        if(!X_lin->empty()) { //don't call if no linear term
            this->checkCudaErrors(X_lin->copyTo(stream, block->trials[mm]->X_lin, true, copyOffset), "GPUGMLMPop_dataset_GPU errors: could not copy X_lin to device!");
        }
    } 

    //upload vectors to GPU
    this->checkCudaErrors(normalizingConstants_trial->copyHostToGPU(stream), "GPUGMLMPop_dataset_GPU errors: could not copy normalizingConstants_trial to device!");
   
    this->checkCudaErrors(ridx_t_all->copyHostToGPU(stream), "GPUGMLMPop_dataset_GPU errors: could not copy ridx_t_all to device!");
    this->checkCudaErrors(id_t_trial->copyHostToGPU(stream), "GPUGMLMPop_dataset_GPU errors: could not copy id_t_trial to device!");
    this->checkCudaErrors(id_a_trialM->copyHostToGPU(stream), "GPUGMLMPop_dataset_GPU errors: could not copy id_a_trialM to device!");
     
    this->checkCudaErrors(dim_N->copyHostToGPU(stream), "GPUGMLMPop_dataset_GPU errors: could not copy dim_N to device!");

    //setup compute space
     LL = new GPUData<FPTYPE>(ce, GPUData_HOST_NONE, stream, dim_N_total(), dim_P());
    this->checkCudaErrors(ce, "GPUGMLMPop_dataset_GPU errors: could not allocate LL on device!");
    dLL = new GPUData<FPTYPE>(ce, GPUData_HOST_NONE, stream, dim_N_total(), dim_P());
    this->checkCudaErrors(ce, "GPUGMLMPop_dataset_GPU errors: could not allocate dLL on device!");

    ridx_sa_all = new GPUData<unsigned int>(ce, GPUData_HOST_NONE, stream, dim_N_total());
    this->checkCudaErrors(ce, "GPUGMLMPop_dataset_GPU errors: could not allocate ridx_sa_all on device!");
    ridx_a_all = new GPUData<unsigned int>(ce, GPUData_HOST_NONE, stream, dim_N_total(), 0);
    this->checkCudaErrors(ce, "GPUGMLMPop_dataset_GPU errors: could not allocate ridx_a_all on device!");
    ridx_st_sall = new GPUData<unsigned int>(ce, GPUData_HOST_PAGELOCKED, stream, dim_M());
    this->checkCudaErrors(ce, "GPUGMLMPop_dataset_GPU errors: could not allocate ridx_st_sall on device!");
    
    lambda = new GPUData<FPTYPE>(ce, GPUData_HOST_NONE, stream, dim_N_total(), dim_P(), dim_J());
    this->checkCudaErrors(ce, "GPUGMLMPop_dataset_GPU errors: could not allocate lambda on device!");

    X_lin_temp = new GPUData<FPTYPE>(ce, GPUData_HOST_NONE, stream, max_trial_length * max_trials_for_sparse_run, dim_B(), X_lin_depth);
    this->checkCudaErrors(ce, "GPUGMLMPop_dataset_GPU errors: could not allocate X_lin_temp on device!");

    dW_trial = new GPUData<FPTYPE>(ce, GPUData_HOST_NONE, stream, dim_M(), dim_P());
    this->checkCudaErrors(ce, "GPUGMLMPop_dataset_GPU errors: could not allocate dW_trial on device!");

    //setup the groups
    for(unsigned int jj = 0; jj < dim_J(); jj++) {
        Groups[jj] = new GPUGMLMPop_dataset_Group_GPU<FPTYPE>(jj, GMLMPopstructure->Groups[jj], block->trials, this, stream, cusparseHandle_Groups[jj]);
    }
}

template <class FPTYPE>
GPUGMLMPop_dataset_Group_GPU<FPTYPE>::GPUGMLMPop_dataset_Group_GPU(const int groupNum_, const GPUGMLM_structure_Group_args<FPTYPE> * GMLMPopGroupStructure, const std::vector<GPUGMLM_trial_args <FPTYPE> *> trials, const GPUGMLMPop_dataset_GPU<FPTYPE> * parent_, const hipStream_t stream, const hipsparseHandle_t & cusparseHandle) : parent(parent_), groupNum(groupNum_) {
    this->dev  = parent->dev;
    this->msg = parent->msg;
    this->switchToDevice();
    hipError_t ce;
    
    //sets up dimensions
    X.resize( GMLMPopGroupStructure->dim_D(msg));
    XF.resize(dim_D());
    iX.resize(dim_D());
    X_temp.resize( dim_D());
    lambda_d.resize( dim_D());

    isShared = new GPUData<bool>(ce, GPUData_HOST_STANDARD, stream, dim_D()); 
    this->checkCudaErrors(ce, "GPUGMLMPop_dataset_Group_GPU errors: could not allocate isShared!");
    isSharedIdentity = new GPUData<bool>(ce, GPUData_HOST_STANDARD, stream, dim_D()); 
    this->checkCudaErrors(ce, "GPUGMLMPop_dataset_Group_GPU errors: could not allocate isSharedIdentity!");

    dim_A = GMLMPopGroupStructure->dim_A;

    size_t dim_T_total = 1;
    std::vector<size_t> dim_F_c;
    dim_F_c.assign(dim_D(), 1);
    size_t max_dim_F_dim_P = parent->dim_P();
    for(unsigned int ss = 0; ss < GMLMPopGroupStructure->dim_S(); ss++) {
        dim_T_total *= GMLMPopGroupStructure->dim_T[ss];
        dim_F_c[GMLMPopGroupStructure->factor_idx[ss]] *= GMLMPopGroupStructure->dim_T[ss];

        max_dim_F_dim_P = max(max_dim_F_dim_P,  dim_F_c[GMLMPopGroupStructure->factor_idx[ss]]);
    }

    if(GMLMPopGroupStructure->dim_S() == 0 || dim_T_total == 0) {
        this->output_stream << "GPUGMLMPop_dataset_Group_GPU errors: tensor has no components!";
        this->msg->callErrMsgTxt(output_stream);
    }
    if(GMLMPopGroupStructure->dim_A == 0) {
        this->output_stream << "GPUGMLMPop_dataset_Group_GPU errors: tensor has no events/data!";
        this->msg->callErrMsgTxt(output_stream);
    }
    if(GMLMPopGroupStructure->dim_R_max < 1) {
        this->output_stream << "GPUGMLMPop_dataset_Group_GPU errors: tensor max rank must be at least 1!";
        this->msg->callErrMsgTxt(output_stream);
    }
    
    //allocated space for regressors and copy to GPU
    size_t max_dim_X_shared = parent->dim_N_total();

    for(unsigned int dd = 0; dd < dim_D(); dd++) {
        (*isShared)[dd] = !(GMLMPopGroupStructure->X_shared[dd]->empty());

        if((*isShared)[dd]) {
            //if shared
            max_dim_X_shared = max(max_dim_X_shared, GMLMPopGroupStructure->X_shared[dd]->getSize(0));

            //gets depth
            size_t depth = GMLMPopGroupStructure->X_shared[dd]->getSize(2);
            if(depth != 1) {
                this->output_stream << "GPUGMLMPop_dataset_Group_GPU errors: X_shared depth must be 1!";
                this->msg->callErrMsgTxt(output_stream);
            }

            //allocate space
            X[dd]  = new GPUData<FPTYPE>(ce, GPUData_HOST_NONE, stream, GMLMPopGroupStructure->X_shared[dd]->getSize(0), dim_F_c[dd], depth);
            iX[dd] = new GPUData<int   >(ce, GPUData_HOST_NONE, stream, parent->dim_N_total(), dim_A);

            //copy to GPU
            this->checkCudaErrors(X[dd]->copyTo(stream, GMLMPopGroupStructure->X_shared[dd], false), "GPUGMLMPop_dataset_Group_GPU errors: could not copy X[dd] shared to device!");
       
            // copy each trial's data to GPU
            for(unsigned int mm = 0; mm < trials.size(); mm++) {
                hipPos copyOffset = make_hipPos((*(parent->ridx_t_all))[mm], 0, 0); //get row for current trial
                this->checkCudaErrors(iX[dd]->copyTo(stream, trials[mm]->Groups[groupNum]->iX[dd], true, copyOffset), "GPUGMLMPop_dataset_Group_GPU errors: could not copy iX[dd] shared to device!");
            }

            //check if X_shared is the identity matrix
            if(X[dd]->getSize(0) == X[dd]->getSize(1)) {
                (*isSharedIdentity)[dd] = true;
                for(unsigned int ii = 0; ii < X[dd]->getSize(0) && (*isSharedIdentity)[dd]; ii++) {
                    for(unsigned int jj = 0; jj < X[dd]->getSize(1) && (*isSharedIdentity)[dd]; jj++) {
                        if(ii == jj) {
                            (*isSharedIdentity)[dd] = 1 == (*(GMLMPopGroupStructure->X_shared[dd]))(ii,jj);
                        }
                        else {
                            (*isSharedIdentity)[dd] = 0 == (*(GMLMPopGroupStructure->X_shared[dd]))(ii,jj);
                        }
                    }
                }
            }
            else {
                (*isSharedIdentity)[dd] = false;
            }

            if(!((*isSharedIdentity)[dd])) {
                //XF comp space
                XF[dd] = new GPUData<FPTYPE>(ce, GPUData_HOST_NONE, stream, dim_X(dd), GMLMPopGroupStructure->dim_R_max);
                this->checkCudaErrors(ce, "GPUGMLMPop_dataset_Group_GPU errors: could not allocate space for XF[dd] shared!" );
            }
            else {
                XF[dd] = new GPUData<FPTYPE>(ce, GPUData_HOST_NONE, stream, dim_X(dd), GMLMPopGroupStructure->dim_R_max, 0); // is empty, but has correct dimensions
                this->checkCudaErrors(ce, "GPUGMLMPop_dataset_Group_GPU errors: could not allocate space for XF[dd] shared+identity!" );
            }

            //X space for sparse runs
            X_temp[dd] = new GPUData<FPTYPE>(ce, GPUData_HOST_NONE, stream, parent->max_trial_length * parent->max_trials_for_sparse_run, dim_F_c[dd], dim_A, true);
            this->checkCudaErrors(ce, "GPUGMLMPop_dataset_Group_GPU errors: could not allocate space for X_temp[dd]!" );
        }
        else {
            //if local
            (*isSharedIdentity)[dd] = false;

            //gets depth
            size_t depth = trials[0]->Groups[groupNum]->X[dd]->getSize(2);
            if(depth != 1 && depth != dim_A) {
                this->output_stream << "GPUGMLMPop_dataset_Group_GPU errors: X_local depth must be dim_A or 1!";
                this->msg->callErrMsgTxt(output_stream);
            }

            //allocate space
            X[dd]  = new GPUData<FPTYPE>(ce, GPUData_HOST_NONE, stream, parent->dim_N_total(), dim_F_c[dd], depth, true);
            iX[dd] = new GPUData<int   >(ce, GPUData_HOST_NONE, stream, 0, GMLMPopGroupStructure->dim_A);

            // copy each trial's data
            for(unsigned int mm = 0; mm < trials.size(); mm++) {
                hipPos copyOffset = make_hipPos((*(parent->ridx_t_all))[mm], 0, 0); //get row for current trial
                this->checkCudaErrors(X[dd]->copyTo(stream, trials[mm]->Groups[groupNum]->X[dd], true, copyOffset), "GPUGMLMPop_dataset_Group_GPU errors: could not copy X[dd] local to device!");
            }

            //XF comp space
            XF[dd] = new GPUData<FPTYPE>(ce, GPUData_HOST_NONE, stream, parent->dim_N_total(), GMLMPopGroupStructure->dim_R_max, depth, true);
            this->checkCudaErrors(ce, "GPUGMLMPop_dataset_Group_GPU errors: could not allocate space for XF[dd] shared!" );

            //X space for sparse runs
            X_temp[dd] = new GPUData<FPTYPE>(ce, GPUData_HOST_NONE, stream, parent->max_trial_length * parent->max_trials_for_sparse_run, dim_F_c[dd], depth, true);
            this->checkCudaErrors(ce, "GPUGMLMPop_dataset_Group_GPU errors: could not allocate space for X_temp[dd]!" );
        }

    }

    this->checkCudaErrors(isShared->copyHostToGPU(stream), "GPUGMLMPop_dataset_Group_GPU errors: could not copy isShared to device!");
    this->checkCudaErrors(isSharedIdentity->copyHostToGPU(stream), "GPUGMLMPop_dataset_Group_GPU errors: could not copy isSharedIdentity to device!");
    
    //setup compute space
    lambda_v = new GPUData<FPTYPE>(ce, GPUData_HOST_NONE, stream, parent->dim_N_total(), dim_R_max());
    this->checkCudaErrors(ce, "GPUGMLMPop_dataset_Group_GPU errors: could not allocate space for lambda_v!" );

    // pitched memory for lambda_d: note arrangement is (dim_N_total*dim_A) x dim_R
    //                                this stacks the events to line up with X or S
    lambda_d.assign(dim_D(), NULL);
    for(unsigned int dd = 0; dd < dim_D(); dd++) {
        size_t depth = dim_A;
        if(!((*isShared)[dd]) && X[dd]->getSize(2) == 1) {
            depth = 1;
        }
        lambda_d[dd] = new GPUData<FPTYPE>(ce, GPUData_HOST_NONE, stream, parent->dim_N_total(), dim_R_max(), depth, true);
        this->checkCudaErrors(ce, "GPUGMLMPop_dataset_Group_GPU errors: could not allocate space for lambda_d!" );
    }

    phi_d =  new GPUData<FPTYPE>(ce, GPUData_HOST_NONE, stream, max_dim_X_shared, dim_R_max());
    this->checkCudaErrors(ce, "GPUGMLMPop_dataset_Group_GPU errors: could not allocate space for phi_d!" );

    //setup sparse matrices for dT
    spi_rows.assign(dim_D(), NULL);
    spi_cols.assign(dim_D(), NULL);
    spi_data.assign(dim_D(), NULL);

    spi_S.assign(dim_D(), NULL);
    spi_phi_d.assign(dim_D(), NULL);
    spi_lambda_d.assign(dim_D(), NULL);

    spi_buffer.assign(dim_D(), NULL);
    spi_buffer_size.assign(dim_D(), 0);

    for(unsigned int dd = 0; dd < dim_D(); dd++) {
        if((*isShared)[dd]) {
            //gets the rows and cols of the spm in the correct order
                //shorter algorithm is too slow for my level of patience, so we do this in a couple steps
                //first, get valid entries and number of entries per row of spi_S
            size_t ctr = 0;
            std::vector<int> row_ctr;
            row_ctr.resize(dim_X(dd));
            for(unsigned int mm = 0; mm < parent->dim_M(); mm++) { //for each trial
                for(unsigned int aa = 0; aa < dim_A; aa++) { //for each event
                    for(unsigned int nn = 0; nn < trials[mm]->dim_N(msg); nn++) { //for each observation
                        //gets the entry in the input data
                        int row = (*(trials[mm]->Groups[groupNum]->iX[dd]))(nn, aa);
                        if(row >= 0 && row < dim_X(dd)) { //if valid row (invalid indices are 0's)
                            row_ctr[row]++;
                            ctr++;
                        }
                    }
                }
            }

                //gets to cumulative sum of the rows
            std::vector<int> row_idx;
            row_idx.resize(dim_X(dd));
            row_idx[0] = 0;
            for(unsigned int xx = 1; xx < dim_X(dd); xx++) {
                row_idx[xx] = row_ctr[xx-1] + row_idx[xx-1]; 
            }
                //goes back through the indices and adds them on
            spi_rows[dd] = new GPUData<int>(ce, GPUData_HOST_STANDARD, stream, ctr, 1, 1);
            this->checkCudaErrors(ce, "GPUGMLMPop_dataset_Group_GPU errors: could not allocate space for spi_rows[dd]!");
            spi_cols[dd] = new GPUData<int>(ce, GPUData_HOST_STANDARD, stream, ctr, 1, 1);
            this->checkCudaErrors(ce, "GPUGMLMPop_dataset_Group_GPU errors: could not allocate space for spi_cols[dd]!");

            row_ctr.assign(dim_X(dd), 0); //reset row counter
            for(unsigned int mm = 0; mm < parent->dim_M(); mm++) { //for each trial
                for(unsigned int aa = 0; aa < dim_A; aa++) { //for each event
                    for(unsigned int nn = 0; nn < trials[mm]->dim_N(msg); nn++) { //for each observation
                        //gets the entry in the input data
                        int row = (*(trials[mm]->Groups[groupNum]->iX[dd]))(nn, aa);
                        if(row >= 0 && row < dim_X(dd)) { //if valid row
                            //inserts element
                            size_t entry_num = row_idx[row] + row_ctr[row];
                            (*(spi_cols[dd]))[entry_num] = (*(parent->ridx_t_all))[mm] + nn + aa * parent->dim_N_total();
                            (*(spi_rows[dd]))[entry_num] = row;

                            row_ctr[row]++;
                        }
                    }
                }
            }

            //copy indices to device
            this->checkCudaErrors(spi_rows[dd]->copyHostToGPU(stream), "GPUGMLMPop_dataset_Group_GPU errors: could not copy spi_rows[dd] to device!");
            this->checkCudaErrors(spi_cols[dd]->copyHostToGPU(stream), "GPUGMLMPop_dataset_Group_GPU errors: could not copy spi_cols[dd] to device!");
            
            spi_data[dd] = new GPUData<FPTYPE>(ce, GPUData_HOST_NONE, stream, spi_rows[dd]->size(), 1, 1);
            this->checkCudaErrors(ce, "GPUGMLMPop_dataset_Group_GPU errors: could not allocate space for spi_data[dd]!");

            //setup sparse matrix handle
            hipsparseStatus_t cusparse_stat;
            spi_S[dd] =  new hipsparseSpMatDescr_t;
            cusparse_stat = hipsparseCreateCoo(spi_S[dd],
                        dim_X(dd), lambda_d[dd]->getSize(0) * lambda_d[dd]->getSize(2), //num rows, cols
                        spi_rows[dd]->size(), //number of non-zeros
                        spi_rows[dd]->getData_gpu(), //row offsets
                        spi_cols[dd]->getData_gpu(), //col offsets
                        spi_data[dd]->getData_gpu(), //the entries
                        HIPSPARSE_INDEX_32I,
                        HIPSPARSE_INDEX_BASE_ZERO,
                        getCudaType<FPTYPE>());
            this->checkCudaErrors(cusparse_stat, "GPUGMLMPop_dataset_Group_GPU errors: creating sparse mat spi_S for dT failed.");

            //setup dense handle for lambda_d
            spi_lambda_d[dd] = new hipsparseDnVecDescr_t;
            cusparse_stat = hipsparseCreateDnVec(spi_lambda_d[dd],
                                                lambda_d[dd]->getSize(0) * lambda_d[dd]->getSize(2),  //size
                                                lambda_d[dd]->getData_gpu(),
                                                getCudaType<FPTYPE>());
            output_stream << "GPUGMLMPop_dataset_Group_GPU errors: creating dense vec cusparse handle spi_lambda_d failed.\n";
            lambda_d[dd]->printInfo(output_stream);
            output_stream << " group " << groupNum << "\n";
            output_stream << " factor " << dd << "\n";
            this->checkCudaErrors(cusparse_stat, output_stream.str().c_str());
            output_stream.str("");
            output_stream.clear();

            //setup dense handle for phi_d
            spi_phi_d[dd] = new hipsparseDnVecDescr_t;
            cusparse_stat = hipsparseCreateDnVec(spi_phi_d[dd],
                                                dim_X(dd), //size
                                                phi_d->getData_gpu(), //values
                                                getCudaType<FPTYPE>()); //valueType
            this->checkCudaErrors(cusparse_stat, "GPUGMLMPop_dataset_Group_GPU errors: creating dense vec cusparse handle spi_phi_d failed.");

            //checks buffer for spi
            size_t buffer;
            FPTYPE alpha = 1;
            FPTYPE beta  = 0;
            cusparse_stat = hipsparseSpMV_bufferSize(cusparseHandle,
                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                    &alpha,
                    *(spi_S[dd]),
                    *(spi_lambda_d[dd]),
                    &beta,
                    *( spi_phi_d[dd] ),
                    getCudaType<FPTYPE>(),
                    HIPSPARSE_SPMV_COO_ALG1,//HIPSPARSE_SPMV_ALG_DEFAULT,
                    &(buffer));
            this->checkCudaErrors(cusparse_stat, "GPUGMLMPop_dataset_Group_GPU errors: getting buffer size for SpMV failed.");

            spi_buffer[dd] = new GPUData<char>(ce, GPUData_HOST_NONE, stream, buffer, 1, 1);
            this->checkCudaErrors(ce, "GPUGMLMPop_dataset_Group_GPU errors: could not allocate space for spi_buffer[dd]!" );
            spi_buffer_size[dd] = buffer; 
        }
    }

    this->checkCudaErrors(hipEventCreate(&group_LL_event), "GPUGMLMPop_dataset_Group_GPU errors: could not create LL event!");
}

// destructor
template <class FPTYPE>
GPUGMLMPop_dataset_GPU<FPTYPE>::~GPUGMLMPop_dataset_GPU() {
    cudaSafeFree(Y, "GPUGMLMPop_dataset_GPU errors: could not free Y");
    
    cudaSafeFree(X_lin, "GPUGMLMPop_dataset_GPU errors: could not free X_lin");
    cudaSafeFree(X_lin_temp, "GPUGMLMPop_dataset_GPU errors: could not free X_lin_temp");
    
    cudaSafeFree(normalizingConstants_trial, "GPUGMLMPop_dataset_GPU errors: could not free normalizingConstants_trial");
    
    cudaSafeFree(ridx_t_all   , "GPUGMLMPop_dataset_GPU errors: could not free ridx_t_all");
    cudaSafeFree(ridx_st_sall , "GPUGMLMPop_dataset_GPU errors: could not free ridx_st_sall");
    cudaSafeFree(ridx_sa_all  , "GPUGMLMPop_dataset_GPU errors: could not free ridx_sa_all");
    cudaSafeFree(ridx_a_all  , "GPUGMLMPop_dataset_GPU errors: could not free ridx_a_all");
    
    cudaSafeFree(id_t_trial , "GPUGMLMPop_dataset_GPU errors: could not free id_t_trial");
    cudaSafeFree(id_a_trialM, "GPUGMLMPop_dataset_GPU errors: could not free id_a_trialM");
    
    cudaSafeFree(dim_N, "GPUGMLMPop_dataset_GPU errors: could not free dim_N");
    
    cudaSafeFree( LL, "GPUGMLMPop_dataset_GPU errors: could not free  LL");
    cudaSafeFree(dLL, "GPUGMLMPop_dataset_GPU errors: could not free dLL");
    cudaSafeFree(lambda, "GPUGMLMPop_dataset_GPU errors: could not free lambda");
    cudaSafeFree(dW_trial, "GPUGMLMPop_dataset_GPU errors: could not free dW_trial");

    //clear the groups
    for(auto gg : Groups) {
        delete gg;
    }
}

template <class FPTYPE>
GPUGMLMPop_dataset_Group_GPU<FPTYPE>::~GPUGMLMPop_dataset_Group_GPU() {
    this->checkCudaErrors(hipEventDestroy(group_LL_event), "GPUGMLMPop_dataset_Group_GPU errors: could not clear LL event!");
    cudaSafeFreeVector(X, "GPUGMLMPop_dataset_Group_GPU errors: could not free X[dd]");
    cudaSafeFreeVector(XF, "GPUGMLMPop_dataset_Group_GPU errors: could not free iX[dd]");
    cudaSafeFreeVector(iX, "GPUGMLMPop_dataset_Group_GPU errors: could not free iX[dd]");
    cudaSafeFreeVector(X_temp   , "GPUGMLMPop_dataset_Group_GPU errors: could not free X_temp[dd]");
    
    cudaSafeFree(isShared, "GPUGMLMPop_dataset_Group_GPU errors: could not free isShared");
    cudaSafeFree(isSharedIdentity, "GPUGMLMPop_dataset_Group_GPU errors: could not free isSharedIdentity");

    cudaSafeFree(lambda_v, "GPUGMLMPop_dataset_Group_GPU errors: could not free lambda_v");
    cudaSafeFreeVector(lambda_d, "GPUGMLMPop_dataset_Group_GPU errors: could not free lambda_d[dd]");
    cudaSafeFree(   phi_d, "GPUGMLMPop_dataset_Group_GPU errors: could not free phi_d");

    cudaSafeFreeVector(spi_rows, "GPUGMLMPop_dataset_Group_GPU errors: could not free spi_rows");
    cudaSafeFreeVector(spi_cols, "GPUGMLMPop_dataset_Group_GPU errors: could not free spi_cols");
    cudaSafeFreeVector(spi_data, "GPUGMLMPop_dataset_Group_GPU errors: could not free spi_data");
    cudaSafeFreeVector(spi_buffer, "GPUGMLMPop_dataset_Group_GPU errors: could not free spi_buffer");
    //destroy any cusparse handles
    for(unsigned int dd = 0; dd < spi_S.size(); dd++) {
        if(spi_S[dd] != NULL) {
            this->checkCudaErrors(hipsparseDestroySpMat(*spi_S[dd]), "GPUGMLMPop_dataset_Group_GPU errors: CUSPARSE failed to destroy spi_S descr.");
            delete spi_S[dd];
        }
        if(spi_phi_d[dd] != NULL) {
            this->checkCudaErrors(hipsparseDestroyDnVec(*spi_phi_d[dd]), "GPUGMLMPop_dataset_Group_GPU errors: CUSPARSE failed to destroy spi_phi_d descr.");
        	delete spi_phi_d[dd];
        }
        if(spi_lambda_d[dd] != NULL) {
            this->checkCudaErrors(hipsparseDestroyDnVec(*spi_lambda_d[dd]), "GPUGMLMPop_dataset_Group_GPU errors: CUSPARSE failed to destroy spi_lambda_d descr.");
            delete spi_lambda_d[dd];
        }
    }
}

//=============================================================================================================================================================
//=============================================================================================================================================================
//=============================================================================================================================================================
/*Kernel for each observation in a sparse run, for a group
 * Builds the dense regressor matrix with local regressors
*  ridx_sa_all must be assigned
*/
template <class FPTYPE>
__global__ void kernel_getGroupX_local_full_pop(GPUData_kernel<FPTYPE> X_temp, const GPUData_kernel<FPTYPE> X,
                                    const GPUData_kernel<unsigned int> ridx_sa_all) {
    //get current observation number
    unsigned int tt_start = blockIdx.y * blockDim.y;
    unsigned int aa_start = blockIdx.z * blockDim.z;
    size_t row_start = blockIdx.x * blockDim.x;
    for(size_t row_0 = row_start; row_0 < ridx_sa_all.x; row_0 += blockDim.x * gridDim.x) {
        size_t row = row_0 + threadIdx.x;
        size_t iX_row;
        if(row < ridx_sa_all.x  ) {
            iX_row = ridx_sa_all[row];
        }
        __syncwarp();
        for(size_t aa_0 = aa_start; aa_0 < X_temp.z; aa_0 += blockDim.z * gridDim.z) {
            size_t aa = aa_0 + threadIdx.z;

            for(size_t tt_0 = tt_start; tt_0 < X_temp.y; tt_0 += blockDim.y * gridDim.y) {
                size_t tt = tt_0 + threadIdx.y;

                //for each event 
                if(row < ridx_sa_all.x && tt < X_temp.y ) {
                    X_temp(row, tt, aa) = X(iX_row, tt, aa);
                }
                __syncwarp();
            }
        }
    }
}

//functions to multiply the tensor coefficients by the current parameters
template <class FPTYPE>
void GPUGMLMPop_dataset_Group_GPU<FPTYPE>::multiplyCoefficients(const bool isSparseRun, const bool update_weights, const GPUGMLM_parameters_Group_GPU<FPTYPE> * params, const hipStream_t stream, const hipblasHandle_t cublasHandle, hipEvent_t & paramsLoaded) {
    this->checkCudaErrors(set_dim_R(params->dim_R(), stream), "GPUGMLMPop_dataset_Group_GPU errors: could not set dim_R!");
    if(params->dim_R() == 0) {
        return;
    }
    if(params->dim_R() > dim_R_max()) {
        this->output_stream << "GPUGMLMPop_dataset_Group_GPU errors: dim_R too large for pre-allocated space!";
        this->msg->callErrMsgTxt(output_stream);
    }
    this->checkCudaErrors(hipStreamWaitEvent(stream, paramsLoaded, 0), "GPUGMLMPop_dataset_Group_GPU::multiplyCoefficients errors: could not wait for event.");

    if(isSparseRun && update_weights) {
        this->checkCudaErrors(lambda_v->resize(stream, parent->dim_N_temp, -1, -1), "GPUGMLM_dataset_Group_GPU::multiplyCoefficients errors: could not set size for sparse runs.");
    }
    else if(update_weights) {
        this->checkCudaErrors(lambda_v->resize(stream, parent->lambda->getSize_max(0), -1, -1), "GPUGMLM_dataset_Group_GPU::multiplyCoefficients errors: could not set size for sparse runs.");
    }
    for(unsigned int dd = 0; dd < dim_D(); dd++) {
        GPUData<FPTYPE> * X_c = X[dd];
        if(isSparseRun && update_weights) {
            this->checkCudaErrors(X_temp[dd]->resize(  stream, parent->dim_N_temp, -1, -1), "GPUGMLMPop_dataset_Group_GPU::multiplyCoefficients errors: could not set size for sparse runs.");
            this->checkCudaErrors(lambda_d[dd]->resize(stream, parent->dim_N_temp, -1, -1), "GPUGMLMPop_dataset_Group_GPU::multiplyCoefficients errors: could not set size for sparse runs.");
        }
        else if(update_weights) {
            this->checkCudaErrors(lambda_d[dd]->resize(stream, lambda_d[dd]->getSize_max(0),-1,-1), "GPUGMLMPop_dataset_Group_GPU::multiplyCoefficients errors: could not set size for full runs.");
        }
        if((*isSharedIdentity)[dd]) {
            continue;
        }

        if(isSparseRun && !(*isShared)[dd]) {
            // if sparse run and local regressors, build matrix then multiply
            if(update_weights) {
                dim3 block_size;
                if(dim_F(dd) > 8) { 
                    block_size.y = 8;
                }
                else if(dim_F(dd) >= 4) { 
                    block_size.y = 4;
                }
                block_size.y = 1;
                block_size.x = 1024 / block_size.y;
                dim3 grid_size;
                grid_size.x = parent->dim_N_temp / block_size.x + ((parent->dim_N_temp  % block_size.x == 0)? 0:1);
                grid_size.y = 1;
                kernel_getGroupX_local_full_pop<<<grid_size, block_size, 0, stream>>>(X_temp[dd]->device(), X[dd]->device(), 
                                            parent->ridx_sa_all->device());
                this->checkCudaErrors("GPUGMLMPop_dataset_Group_GPU::multiplyCoefficients errors:  kernel_getGroupX_local_full launch failed");
            }
            X_c = X_temp[dd];
        }

        this->checkCudaErrors(XF[dd]->resize(stream, X_c->getSize(0)), "GPUGMLMPop_dataset_Group_GPU::multiplyCoefficients errors: could not set matrix size for run.");    
        hipblasStatus_t cse = X_c->GEMM(XF[dd], params->F[dd], cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N);
        if(cse != HIPBLAS_STATUS_SUCCESS) {
            this->output_stream << " dd " << dd << "\n";
            X_c->printInfo(output_stream, "X_c");
            XF[dd]->printInfo(output_stream, "XF[dd]");
            params->F[dd]->printInfo(output_stream, "params->F[dd]");
            this->msg->printMsgTxt(output_stream);
        }       
        this->checkCudaErrors(cse, "GPUGMLMPop_dataset_Group_GPU::multiplyCoefficients errors:  X*F -> XF failed");
    }
}

//=============================================================================================================================================================
//=============================================================================================================================================================
//=============================================================================================================================================================

        /*Kernel for each observation, for a group
 * For each component (rr = 0:(dim_R-1)) takes the product of the XT terms into lambda_v, then lambda_v'*V -> lambda 
 * Returns the observation-wise constribution to the rate from this group (lambda) and sets up the dV computation
 *
 * If computing any dT values AND dim_S > 1, needs some dynamic  memory to make this work on both 1080 and 2080 cards well. Memory size in bytes is dim_S * blockDim.x * sizeof(FPTYPE)
 */
template <class FPTYPE, unsigned int max_order>
__global__ void kernel_getGroupRate_pop(GPUData_kernel<FPTYPE> lambda_v, 
        GPUData_array_kernel<FPTYPE,MAX_DIM_D> lambda_d,
        const GPUData_array_kernel<FPTYPE,MAX_DIM_D> XF,
        const GPUData_array_kernel<FPTYPE,MAX_DIM_D> F,
        const GPUData_array_kernel<int,MAX_DIM_D> iX,
        const GPUData_kernel<bool> isShared,
        const GPUData_kernel<bool> isSharedIdentity,
        const GPUData_kernel<unsigned int> id_a_trialM,
        const GPUData_kernel<FPTYPE> trial_weights, 
        const bool compute_dV, const GPUData_kernel<bool> compute_dF, const bool compute_dT_any,
        const GPUData_kernel<unsigned int> ridx_sa_all, const size_t dim_A) {
    __shared__ bool isShared_local[max_order];
    __shared__ bool isSharedIdentity_local[max_order];
    FPTYPE t_array[max_order];

    if(threadIdx.x < XF.N) {
        isShared_local[threadIdx.x] = isShared[threadIdx.x];
        isSharedIdentity_local[threadIdx.x] = isSharedIdentity[threadIdx.x];
    }
    int idx_0 = -1;
    __syncthreads();

    for(size_t row_0 = blockIdx.x * blockDim.x; row_0 < lambda_v.x; row_0 += blockDim.x * gridDim.x) {
        size_t row = row_0 + threadIdx.x;
        size_t iX_row = row; //if full run
        if(ridx_sa_all.y > 0 && row < ridx_sa_all.x) {
            //if sparse run
            iX_row = ridx_sa_all[row];
        }
        bool rowIncluded = true;
        if( row >= lambda_v.x || !(trial_weights.y == 0 || trial_weights.y > 1 || (trial_weights.y == 1 && trial_weights[id_a_trialM[iX_row]] != 0))) {
            rowIncluded = false;
        }
        __syncthreads();

        //for each rank
        for(unsigned int rr_0 = blockIdx.y * blockDim.y; rr_0 < lambda_v.y; rr_0 += blockDim.y * gridDim.y) { //dim_R = V->Y
            unsigned int rr = rr_0 + threadIdx.y;
            bool elementIncluded = rowIncluded && rr < lambda_v.y;

            FPTYPE lv = 0;
            for(unsigned int aa = 0; aa < dim_A; aa++) { //over dim_A
                //for each event 
                FPTYPE lv_aa = 1;
                //for each factor
                for(unsigned int dd = 0; dd < XF.N; dd++) { //dim_D = XF->N, dim_S = T->N

                    if(elementIncluded && isShared_local[dd]) { //if trial not censored
                        idx_0 = iX[dd](iX_row, aa); //get index for shared regressors
                                                    // this is pulled up too many times (is same over all rr): haven't found a good way to parallelize
                    }
                    __syncwarp(); 
                    if(elementIncluded && isSharedIdentity_local[dd]) { 
                        if(idx_0 >= 0 && idx_0 < F[dd].x) {
                            t_array[dd]  = F[dd](idx_0, rr);
                        }
                        else {
                            t_array[dd] = 0;
                        }
                    }
                    __syncwarp(); 
                    if(elementIncluded && (!isShared_local[dd] || !isSharedIdentity_local[dd])) { 
                        if(!isShared_local[dd]) {
                            t_array[dd]  = XF[dd](row, rr, aa);
                        }
                        else if(idx_0 >= 0 && idx_0 < XF[dd].x) {
                            t_array[dd]  = XF[dd](idx_0, rr);
                        }
                        else {
                            t_array[dd] = 0;
                        }
                    }
                    lv_aa *= t_array[dd] ;
                    __syncwarp(); 
                } // dd
                lv += lv_aa;

                //sets up any dT matrices (doing this here eliminates the need to go back through the XT matrices in a different kernel)
                //  I do this outside the previous loop because otherwise everything was super slow on the 1080 cards
                if(compute_dT_any) {
                    for(unsigned int dd = 0 ; dd < XF.N; dd++) {
                        if(elementIncluded && compute_dF[dd]) {
                            FPTYPE tt = 1;
                            for(unsigned int dd2 = 0; dd2 < XF.N; dd2++) {
                                if(dd2 != dd) {
                                    tt *= t_array[dd2];
                                }
                            }
                            if(aa < lambda_d[dd].z) {
                                lambda_d[dd](row, rr, aa) = tt;
                            }
                            else {
                                lambda_d[dd](row, rr, aa) += tt;
                            }
                        }
                        __syncwarp();
                    } // dd
                }
                __syncthreads();
            }// aa
            if(elementIncluded) {
                lambda_v(row, rr) = lv;
            }
            __syncthreads();
        } // rr
    } // row
}

template <class FPTYPE>
void GPUGMLMPop_dataset_Group_GPU<FPTYPE>::getGroupRate(const bool isSparseRun, const GPUGMLM_parameters_Group_GPU<FPTYPE> * params, const GPUGMLM_group_computeOptions * opts, const hipStream_t stream, const hipblasHandle_t cublasHandle) { 
    if(params->dim_R() == 0) {
        // set lambda to 0
        FPTYPE * col = parent->lambda->getData_gpu() + groupNum * parent->lambda->getLD_gpu();
        this->checkCudaErrors(hipMemsetAsync(col, 0, parent->lambda->getSize(0)*sizeof(FPTYPE), stream), "GPUGMLMPop_dataset_Group_GPU::getGroupRate errors: errors setting rate to 0 for dim_R=0 group");
    }
    else {
        dim3 block_size;
        dim3 grid_size;

        if(dim_R() > 8) {
            block_size.y = 4;
        }
        else  if(dim_R() > 4) {
            block_size.y = 2;
        }
        else {
            block_size.y = 1;
        }
        block_size.x = 1024 / block_size.y;
        size_t max_blocks_needed  = parent->lambda->getSize(0) / block_size.x + ((parent->lambda->getSize(0) % block_size.x == 0)? 0:1);
        size_t blocks_to_use = (parent->dim_J() == 1) ? 1024 : 512;
        grid_size.x  = min(max_blocks_needed, blocks_to_use);
        grid_size.y  = dim_R() / block_size.y + ((dim_R() % block_size.x == 0)? 0:1);

        bool compute_dT_any = false;
        for(unsigned int ss = 0; ss < params->dim_S(); ss++) {
            if(opts->compute_dT[ss]) {
                compute_dT_any = true;
                break;
            }
        }
        
        switch( params->dim_S()) {
                case 1:
                    kernel_getGroupRate_pop<FPTYPE,1><<<grid_size, block_size, 0, stream>>>( lambda_v->device(),  GPUData<FPTYPE>::assembleKernels(lambda_d), 
                                                                                        GPUData<FPTYPE>::assembleKernels(XF),  GPUData<FPTYPE>::assembleKernels(params->F),  GPUData<int>::assembleKernels(iX),
                                                                                        isShared->device(), isSharedIdentity->device(),
                                                                                        parent->id_a_trialM->device(),
                                                                                        params->getTrialWeights()->device(),
                                                                                        opts->compute_dV, params->compute_dF->device(), compute_dT_any,
                                                                                        parent->ridx_a_all_c->device(), dim_A);
                    break;
                case 2:
                    kernel_getGroupRate_pop<FPTYPE,2><<<grid_size, block_size, 0, stream>>>( lambda_v->device(),  GPUData<FPTYPE>::assembleKernels(lambda_d), 
                                                                                        GPUData<FPTYPE>::assembleKernels(XF),  GPUData<FPTYPE>::assembleKernels(params->F),  GPUData<int>::assembleKernels(iX),
                                                                                        isShared->device(), isSharedIdentity->device(),
                                                                                        parent->id_a_trialM->device(),
                                                                                        params->getTrialWeights()->device(),
                                                                                        opts->compute_dV, params->compute_dF->device(), compute_dT_any,
                                                                                        parent->ridx_a_all_c->device(), dim_A);
                    break;
                case 3:
                    kernel_getGroupRate_pop<FPTYPE,3><<<grid_size, block_size, 0, stream>>>( lambda_v->device(),  GPUData<FPTYPE>::assembleKernels(lambda_d), 
                                                                                        GPUData<FPTYPE>::assembleKernels(XF),  GPUData<FPTYPE>::assembleKernels(params->F),  GPUData<int>::assembleKernels(iX),
                                                                                        isShared->device(), isSharedIdentity->device(),
                                                                                        parent->id_a_trialM->device(),
                                                                                        params->getTrialWeights()->device(),
                                                                                        opts->compute_dV, params->compute_dF->device(), compute_dT_any,
                                                                                        parent->ridx_a_all_c->device(), dim_A);
                    break;
                case 4:
                    kernel_getGroupRate_pop<FPTYPE,4><<<grid_size, block_size, 0, stream>>>( lambda_v->device(),  GPUData<FPTYPE>::assembleKernels(lambda_d), 
                                                                                        GPUData<FPTYPE>::assembleKernels(XF),  GPUData<FPTYPE>::assembleKernels(params->F),  GPUData<int>::assembleKernels(iX),
                                                                                        isShared->device(), isSharedIdentity->device(),
                                                                                        parent->id_a_trialM->device(),
                                                                                        params->getTrialWeights()->device(),
                                                                                        opts->compute_dV, params->compute_dF->device(), compute_dT_any,
                                                                                        parent->ridx_a_all_c->device(), dim_A);
                    break;
                case 5:
                    kernel_getGroupRate_pop<FPTYPE,5><<<grid_size, block_size, 0, stream>>>( lambda_v->device(),  GPUData<FPTYPE>::assembleKernels(lambda_d), 
                                                                                        GPUData<FPTYPE>::assembleKernels(XF),  GPUData<FPTYPE>::assembleKernels(params->F),  GPUData<int>::assembleKernels(iX),
                                                                                        isShared->device(), isSharedIdentity->device(),
                                                                                        parent->id_a_trialM->device(),
                                                                                        params->getTrialWeights()->device(),
                                                                                        opts->compute_dV, params->compute_dF->device(), compute_dT_any,
                                                                                        parent->ridx_a_all_c->device(), dim_A);
                    break;
                case 6:
                    kernel_getGroupRate_pop<FPTYPE,6><<<grid_size, block_size, 0, stream>>>( lambda_v->device(),  GPUData<FPTYPE>::assembleKernels(lambda_d), 
                                                                                        GPUData<FPTYPE>::assembleKernels(XF),  GPUData<FPTYPE>::assembleKernels(params->F),  GPUData<int>::assembleKernels(iX),
                                                                                        isShared->device(), isSharedIdentity->device(),
                                                                                        parent->id_a_trialM->device(),
                                                                                        params->getTrialWeights()->device(),
                                                                                        opts->compute_dV, params->compute_dF->device(), compute_dT_any,
                                                                                        parent->ridx_a_all_c->device(), dim_A);
                    break;
                case 7:
                    kernel_getGroupRate_pop<FPTYPE,7><<<grid_size, block_size, 0, stream>>>( lambda_v->device(),  GPUData<FPTYPE>::assembleKernels(lambda_d), 
                                                                                        GPUData<FPTYPE>::assembleKernels(XF),  GPUData<FPTYPE>::assembleKernels(params->F),  GPUData<int>::assembleKernels(iX),
                                                                                        isShared->device(), isSharedIdentity->device(),
                                                                                        parent->id_a_trialM->device(),
                                                                                        params->getTrialWeights()->device(),
                                                                                        opts->compute_dV, params->compute_dF->device(), compute_dT_any,
                                                                                        parent->ridx_a_all_c->device(), dim_A);
                    break;
                default:
                this->checkCudaErrors(hipErrorInvalidConfiguration, "GPUGMLMPop_dataset_Group_GPU::getGroupRate_pop errors:  kernel_getGroupRate_pop launch failed - invalid tensor order");
        }
        this->checkCudaErrors("GPUGMLMPop_dataset_Group_GPU::getGroupRate_pop errors:  kernel_getGroupRate_pop launch failed");

        // multiply lambda_v * V' -> lambda(:, :, groupNum)
        FPTYPE alpha = 1;
        FPTYPE beta  = 0;
        hipblasStatus_t ce =  cublasGEMM(cublasHandle,
                              HIPBLAS_OP_N,
                              HIPBLAS_OP_T,
                              lambda_v->getSize(0), dim_P(), dim_R(),
                              &alpha,
                              lambda_v->getData_gpu(), lambda_v->getLD_gpu(),
                              params->V->getData_gpu(), params->V->getLD_gpu(),
                              &beta,
                              parent->lambda->getData_gpu() + groupNum*parent->lambda->getInc_gpu(), parent->lambda->getLD_gpu());
        this->checkCudaErrors(ce, "GPUGMLMPop_dataset_Group_GPU::getGroupRate errors:  lambda_v * V' -> lambda(:, :, groupNum) failed");
    }
    this->checkCudaErrors(hipEventRecord(group_LL_event, stream), "GPUGMLMPop_dataset_Group_GPU::getGroupRate errors: could not add LL event to stream!");
}

//=============================================================================================================================================================
//=============================================================================================================================================================
//=============================================================================================================================================================

/* Kernel for each entry of the sparse matrix for S*lambda_t -> phi_t (shared regressor compression)
*  sets up the elements of S to be a column of lambda_d
*/
template <class FPTYPE>
__global__ void kernel_set_spi_S_pop( GPUData_kernel<FPTYPE> S,  const GPUData_kernel<FPTYPE> lambda_v,
                               const GPUData_kernel<int> S_idx, const unsigned int col) {
    for(size_t nn = blockIdx.x * blockDim.x + threadIdx.x; nn < S.x; nn += gridDim.x * blockDim.x) {
        S[nn] = lambda_v(S_idx[nn] % lambda_v.x, col);
    }
}

template <class FPTYPE>
__global__ void kernel_PointWiseMultiply_derivativeSetup_pop( GPUData_kernel<FPTYPE> lambda_d,  const GPUData_kernel<FPTYPE> lambda_v) {
    size_t row_start = blockIdx.x * blockDim.x;
    size_t col_start = blockIdx.y * blockDim.y;
    size_t sec_start = blockIdx.z * blockDim.z;
    for(size_t row_0 = row_start; row_0 < lambda_d.x; row_0 += blockDim.x * gridDim.x) {
        size_t row = row_0 + threadIdx.x;
        for(size_t col_0 = col_start; col_0 < lambda_d.y; col_0 += blockDim.y * gridDim.y) {
            size_t col = col_0 + threadIdx.y;
            for(size_t sec_0 = sec_start; sec_0 < lambda_d.z; sec_0 += blockDim.z * gridDim.z) {
                size_t sec = sec_0 + threadIdx.z;
                if(row < lambda_d.x && col < lambda_d.y && sec < lambda_d.z) {
                    lambda_d(row, col, sec) *= lambda_v(row, col);
                }
                __syncwarp();
            }
        }
    }
}

/*Kernel for each observation in a sparse run, for a group
*  ridx_sa_all must be assigned
*/
template <class FPTYPE>
__global__ void kernel_getGroupX_shared_full_pop(GPUData_kernel<FPTYPE> X_temp, const GPUData_kernel<FPTYPE> X,
                                    GPUData_kernel<int> iX,      
                                    GPUData_kernel<unsigned int> ridx_sa_all,
                                    const bool isIdentity)   {
    //get current observation number
    unsigned int tt_start = blockIdx.y * blockDim.y;
    size_t row_start = blockIdx.x * blockDim.x;
    for(size_t row_0 = row_start; row_0 < X_temp.x; row_0 += blockDim.x * gridDim.x) {
        //for each regressor (on this thread)
        for(unsigned int tt_0 = tt_start; tt_0 < X.y; tt_0 += blockDim.y * gridDim.y) {
            size_t row = row_0 + threadIdx.x;
            size_t tt  = tt_0 + threadIdx.y;

            if(row < X_temp.x && tt < X.y) {
                size_t iX_row;
                iX_row = ridx_sa_all[row];
                //for each event 
                for(unsigned int aa = 0; aa < iX.y; aa++) {
                    int idx_0 = iX(iX_row, aa);
                    if(idx_0 < 0 || idx_0 >= X.x) {
                        X_temp(row, tt, aa) = 0;
                    }
                    else {
                        if(isIdentity) {
                            X_temp(row, tt, aa) = (idx_0 == tt) ?  1 : 0;
                        }
                        else {
                            X_temp(row, tt, aa) = X(idx_0, tt);
                        }
                    }
                }
            }
            __syncwarp();
        }
    }
}

template <class FPTYPE>
void GPUGMLMPop_dataset_Group_GPU<FPTYPE>::computeDerivatives(GPUGMLM_results_Group_GPU<FPTYPE> * results, const bool isSparseRun, const bool update_weights, GPUGMLM_parameters_Group_GPU<FPTYPE> * params, const GPUGMLM_group_computeOptions * opts, const hipStream_t stream, const hipblasHandle_t cublasHandle, const hipsparseHandle_t cusparseHandle, hipEvent_t & main_LL_event) {
    if(params->dim_R() == 0) {
        return; //nothing to compute
    }
    this->checkCudaErrors(hipStreamWaitEvent(stream, main_LL_event, 0), "GPUGMLMPop_dataset_Group_GPU::computeDerivatives errors: could not wait for stream");
    
    if(opts->compute_dV) {
        //for each neuron
        this->checkCudaErrors(parent->dLL->GEMM(results->dV, lambda_v, cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N), "GPUGMLMPop_dataset_Group_GPU::computeDerivatives errors:  dLL'*lambda_v -> dV failed");
    }

    //check if computing any derivatives first
    std::vector<bool> compute_dF;
    compute_dF.assign(dim_D(), false);
    for(unsigned int ss = 0; ss < params->dim_S(); ss++) {
        unsigned int dd = (*(params->factor_idx))[ss];
        compute_dF[dd] = compute_dF[dd] || opts->compute_dT[ss];
    }

    // compute lambda_v = dLL * V
    for(unsigned int dd = 0; dd < dim_D(); dd++) {
        if(compute_dF[dd]) {
            this->checkCudaErrors(parent->dLL->GEMM(lambda_v, params->V, cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N), "GPUGMLMPop_dataset_Group_GPU::computeDerivatives errors:  dLL*V -> lambda_v failed");
            break;
        }
    }

    //for each factor
    for(unsigned int dd = 0; dd < dim_D(); dd++) {
        if(compute_dF[dd]) {
            // lambda_d init setup in the kernel call in computeRateParts 
            // two steps
            //  lambda_d = lambda_d .* lambda_v
            //  matrix mult of X'*(lambda_d)
                    
            GPUData<FPTYPE> * phi_c;
            GPUData<FPTYPE> * X_c;
            if((*isShared)[dd] && !isSparseRun) { // only do this if doing full run
                //this step is faster with sparse matrices for shared regressors

                //call kernel to setup entries spi to dLL 
                dim3 block_size;
                block_size.x = 1024;
                dim3 grid_size;
                size_t max_blocks_needed  = spi_rows[dd]->size()/ block_size.x + ((spi_rows[dd]->size() % block_size.x == 0)? 0:1);
                size_t blocks_to_use = (parent->dim_J() == 1) ? 1024 : 512;
                grid_size.x  = min(max_blocks_needed, blocks_to_use);

                FPTYPE alpha = 1;
                FPTYPE beta  = 0;
                for(unsigned int rr = 0; rr < params->dim_R(); rr++) {
                    kernel_set_spi_S_pop<<<grid_size, block_size, 0, stream>>>(spi_data[dd]->device(), lambda_v->device(),
                                                         spi_cols[dd]->device(), rr);
                    this->checkCudaErrors("GPUGMLMPop_dataset_Group_GPU::computeDerivatives errors:  kernel_set_spi_S launch failed");

                    //I found - on a 1080ti at least - doing this series of SpMV ops was typically faster than a single SpMM (annoyingly)
                    hipsparseStatus_t cusparse_stat;
                    cusparse_stat = hipsparseDnVecSetValues(*(spi_lambda_d[dd]), lambda_d[dd]->getData_gpu() + rr*lambda_d[dd]->getLD_gpu());
                    this->checkCudaErrors(cusparse_stat, "GPUGMLMPop_dataset_Group_GPU errors: hipsparseDnVecSetValues failed for lambda_t.");
                    if((*isSharedIdentity)[dd]) {
                        cusparse_stat = hipsparseDnVecSetValues(*(spi_phi_d[dd]), results->dF[dd]->getData_gpu() + rr*results->dF[dd]->getLD_gpu());
                    }
                    else {
                        cusparse_stat = hipsparseDnVecSetValues(*(spi_phi_d[dd]), phi_d->getData_gpu() + rr*phi_d->getLD_gpu());
                    }
                    this->checkCudaErrors(cusparse_stat, "GPUGMLMPop_dataset_Group_GPU errors: hipsparseDnVecSetValues failed for phi_d.");
                       
                    cusparse_stat = hipsparseSpMV(cusparseHandle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha,
                                 *(spi_S[dd]),
                                 *(spi_lambda_d[dd]),
                                 &beta,
                                 *(spi_phi_d[dd]),
                                 getCudaType<FPTYPE>(),
                                 HIPSPARSE_SPMV_COO_ALG1, //HIPSPARSE_SPMV_ALG_DEFAULT,
                                 spi_buffer[dd]->getData_gpu()); //spi_buffer[dd]->getData_gpu()

                    this->output_stream << "GPUGMLMPop_dataset_Group_GPU errors: S*lambda->phi_t SpMV failed.\n";
                    this->output_stream << " rr = " << rr << ", dim_R = " << params->dim_R() << ", dd = " << dd << "\n";
                    this->output_stream << "buffer size " << spi_buffer_size[dd]  << "\n";
                    const char * cc = spi_buffer[dd]->getData_gpu();
                    long long cc2 = reinterpret_cast<long long>(cc);
                    this->output_stream << "buffer address " << cc2   << " (% 128 = " <<  cc2 % 128 << ")" << "\n";

                    this->checkCudaErrors(cusparse_stat, output_stream.str().c_str());

                    this->output_stream.str("");
                    this->output_stream.clear();
                }

                X_c   = X[dd];
                phi_c = phi_d;
            }
            else { 
                if((*isShared)[dd] && update_weights) { 
                    //  if doing sparse run with shared regressor, builds temporary X matrix (local regressors)
                    dim3 block_size;
                    block_size.y = 1;
                    block_size.x = 1024 / block_size.y;
                    dim3 grid_size;
                    size_t max_blocks_needed  = X_temp[dd]->getSize(0)  / block_size.x + ((X_temp[dd]->getSize(0)  % block_size.x == 0)? 0:1);
                    size_t blocks_to_use = (parent->dim_J() == 1) ? 1024 : 512;
                    grid_size.x  = min(max_blocks_needed, blocks_to_use);
                    grid_size.y = 1;

                    kernel_getGroupX_shared_full_pop<<<grid_size, block_size, 0, stream>>>(X_temp[dd]->device(), X[dd]->device(), 
                                                    iX[dd]->device(), 
                                                    parent->ridx_sa_all->device(),
                                                    (*isSharedIdentity)[dd]);
                    this->checkCudaErrors("GPUGMLMPop_dataset_Group_GPU::computeDerivatives errors:  kernel_getGroupX_shared_full launch failed");
                }
                
                // if local regressors
                dim3 block_size;
                if(dim_R() > 8) {
                    block_size.y = 4;
                }
                else  if(dim_R() > 4) {
                    block_size.y = 2;
                }
                else {
                    block_size.y = 1;
                }
                block_size.x = 1024 / block_size.y;
                dim3 grid_size;
                size_t max_blocks_needed = lambda_v->getSize(0) / block_size.x + ((lambda_v->getSize(0) % block_size.x == 0)? 0:1);
                size_t blocks_to_use = (parent->dim_J() == 1) ? 1024 : 512;
                grid_size.x  = min(max_blocks_needed, blocks_to_use);
                
                grid_size.y = dim_R()  / block_size.y + ((dim_R()  % block_size.y == 0)? 0:1);
                        
                kernel_PointWiseMultiply_derivativeSetup_pop<<<grid_size, block_size, 0, stream>>>(lambda_d[dd]->device(), lambda_v->device());
                this->checkCudaErrors("GPUGMLMPop_dataset_Group_GPU::computeDerivatives errors:  kernel_PointWiseMultiply_derivativeSetup launch failed");

                if(isSparseRun) {
                    // if sparse run
                    X_c     = X_temp[dd];
                    phi_c = lambda_d[dd];
                }
                else {
                    // if local regressors and full run
                    X_c   = X[dd];
                    phi_c = lambda_d[dd];
                }
            }

            this->checkCudaErrors(phi_c->resize(stream, X_c->getSize(0), results->dF[dd]->getSize(1), X_c->getSize(2)), "GPUGMLMPop_dataset_Group_GPU::computeDerivatives errors: setting size of phi_c failed");

            // matrix mult to get dF (local and shared)
            if((*isShared)[dd] && !isSparseRun && (*isSharedIdentity)[dd]) {
                //nothing needed
            }
            else {
                //int mt;
                //hipblasStatus_t cse = X_c->GEMM(results->dF[dd], phi_c, cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, 1, 0, NULL, &mt);
                //output_stream << "DT multiplication type " << mt << "\n";
                //this->msg->printMsgTxt(output_stream);
                hipblasStatus_t cse = X_c->GEMM(results->dF[dd], phi_c, cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N);
                this->checkCudaErrors(static_cast<hipError_t>(cse), "GPUGMLMPop_dataset_Group_GPU::computeDerivatives errors:   X'*phi -> dF");
            }
            
            // matrix mults to get dT
            if((*(params->N_per_factor))[dd] > 1) {
                for(unsigned int ss = 0; ss < params->dim_S(); ss++) {
                    if((*(params->factor_idx))[ss] == dd && opts->compute_dT[ss]) {
                        this->checkCudaErrors(params->dF_dT[ss]->GEMVs(results->dT[ss], results->dF[dd], cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N), "GPUGMLMPop_dataset_Group_GPU::computeDerivatives errors: dF_dT'*dF -> dT");
                    }
                }
            }
        }
    }
}

//=============================================================================================================================================================
//=============================================================================================================================================================
//=============================================================================================================================================================

       
//explicitly create classes for single and double precision floating point for library
template class GPUGMLMPop_computeBlock<float>;
template class GPUGMLMPop_computeBlock<double>;

template class GPUGMLMPop_dataset_Group_GPU<float>;
template class GPUGMLMPop_dataset_Group_GPU<double>;
template class GPUGMLMPop_dataset_GPU<float>;
template class GPUGMLMPop_dataset_GPU<double>;
};//namespace